#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "wham.h"

// #define kB 0.008314
#define kB 0.00198614
#define MAXLENGTH 4096
#define BLOCK 256
#define SBLOCK 16

__device__ __host__ inline
double logadd(double lnA,double lnB)
{
  if (lnA>lnB) {
    return lnA+log(1+exp(lnB-lnA));
  } else if (lnA==(-INFINITY)) {
    return lnB;
  } else {
    return lnB+log(1+exp(lnA-lnB));
  }
}

__device__ inline
void atomic_logadd(double *p_lnA,double lnB)
{
  double lnA, lnC;
  double tmp_lnA;
  tmp_lnA=p_lnA[0];
  do {
    lnA=tmp_lnA;
    if (lnA>lnB) {
      lnC=lnA+log(1+exp(lnB-lnA));
    } else if (lnA==(-INFINITY)) {
      lnC=lnB;
    } else {
      lnC=lnB+log(1+exp(lnA-lnB));
    }
    // tmp_lnA=atomicCAS(p_lnA,lnA,lnC);
    // tmp_lnA=__int_as_float(atomicCAS((int*) p_lnA,__float_as_int(lnA),__float_as_int(lnC)));
    tmp_lnA=__longlong_as_double(atomicCAS((unsigned long long int*) p_lnA,__double_as_longlong(lnA),__double_as_longlong(lnC)));
  } while (lnA != tmp_lnA);
}

struct_data* readdata(int argc, char *argv[])
{
  char *Edir,*Ddir;
  FILE *fp,*fpE,*fpQ;
  int i;
  int s1,s2;
  int j,jN;
  int iN;
  int count;
  // int iB0,iB1;
  int B_N;
  int iB0;
  char fnm[MAXLENGTH];
  char line[MAXLENGTH];
  char *linebuffer;
  int ibuffer;
  int n;
  double E,q;
  int B0_MAX=INT_MIN;
  int B0_MIN=INT_MAX;
  // int B1_MAX=INT_MIN;
  // int B1_MIN=INT_MAX;
  struct_data *data;

  data=(struct_data*) malloc(sizeof(struct_data));

  // argv[0] is function name
  if (argc>1) {
    count=sscanf(argv[1],"%d",&(data->Nsim));
    if (count!=1) {
      fprintf(stderr,"Error, argument is not an integer\n");
      exit(1);
    }
  } else {
    fprintf(stderr,"Error, no input. How many potentials are you using?\n");
    exit(1);
  }

  fp=fopen("../nblocks","r");
  if (fp==NULL) {
    fprintf(stderr,"Error, ../nblocks does not exist\n");
    exit(1);
  }
  if (fscanf(fp,"%d",&(data->Nblocks))==0) {
    fprintf(stderr,"Error, could not read number of blocks from ../nblocks\n");
    exit(1);
  }
  fclose(fp);
  fp=fopen("../nsubs","r");
  if (fp==NULL) {
    fprintf(stderr,"Error, ../nsubs does not exist\n");
    exit(1);
  }
  data->Nsites=0;
  while (fscanf(fp,"%d",&i)==1) {
    data->Nsites++;
  }
  fclose(fp);
  data->Nsubs=(int*) malloc(data->Nsites*sizeof(int));
  data->block0=(int*) malloc((data->Nsites+1)*sizeof(int));
  fp=fopen("../nsubs","r");
  data->block0[0]=0;
  for (i=0; i<data->Nsites; i++) {
    fscanf(fp,"%d",&(data->Nsubs[i]));
    data->block0[i+1]=data->block0[i]+data->Nsubs[i];
  }
  fclose(fp);

  if (argc>3) {
    Edir=argv[2];
    Ddir=argv[3];
  } else {
    fprintf(stderr,"Error, missing input and output directory.\n");
    exit(1);
  }

  if (argc>4) {
    sscanf(argv[4],"%d",&(data->ms));
  } else {
    fprintf(stderr,"Error, 4th argument should indicate whether to use multisite parameters.\n");
    exit(1);
  }

  if (argc>5) {
    sscanf(argv[5],"%d",&(data->msprof));
  } else {
    fprintf(stderr,"Error, 5th argument should indicate whether to use multisite profiles.\n");
    exit(1);
  }

  data->NL=data->Nblocks;
  data->NF=data->Nsim;

  data->Ndim=data->Nsim+data->NL+1+2; // E(1) Lambda(6) Energies(Nsim) BinInd(1) ReactCoord(1)

  data->T_h=(double*) malloc(data->NF*sizeof(double));
  data->beta_h=(double*) malloc(data->NF*sizeof(double));
  hipMalloc(&(data->beta_d),data->NF*sizeof(double));

  for (i=0; i<data->NF; i++) {
    data->T_h[i]=298.15;
    data->beta_h[i]=1.0/(kB*data->T_h[i]);
  }
  hipMemcpy(data->beta_d,data->beta_h,data->NF*sizeof(double),hipMemcpyHostToDevice);

  data->beta_t=1.0/(kB*298.15);

  data->B[0].dx=0.1;
  data->B[1].dx=0.002500025;

  data->B2d[0].dx=0.1;
  data->B2d[1].dx=0.0500005;
  data->B2d[2].dx=0.0500005;

  data->n_h=(int*) malloc(data->NF*sizeof(int));
  hipMalloc(&(data->n_d),data->NF*sizeof(int));
  data->ND=0;
  data->NDmax=0;
  data->NDmax+=MAXLENGTH;
  data->D_h=(double*) malloc(data->NDmax*data->Ndim*(sizeof(double)));
  data->i_h=(int*) malloc(data->NDmax*(sizeof(int)));
  data->lnw_h=(double*) malloc(data->NF*(sizeof(double)));

  for (i=0; i<data->NF; i++) {
    sprintf(fnm,"%s/ESim%d.dat",Edir,i+1);
    fpE=fopen(fnm,"r");
    if (fpE==NULL) {
      fprintf(stderr,"Error, energy file %s does not exist\n",fnm);
      exit(1);
    }
    sprintf(fnm,"%s/Lambda%d.dat",Ddir,i+1);
    fpQ=fopen(fnm,"r");
    if (fpQ==NULL) {
      fprintf(stderr,"Error, contact file %s does not exist\n",fnm);
      exit(1);
    }

    data->lnw_h[i]=(data->NF-i-1)*log(1.0);
    // data->lnw_h[i]=(data->NF-i-1)*(-1.0);

    n=0;

    while (fgets(line,MAXLENGTH,fpE) != NULL) {
      if (data->ND>=data->NDmax) {
        data->NDmax+=MAXLENGTH;
        data->D_h=(double*) realloc(data->D_h,data->NDmax*data->Ndim*sizeof(double));
        data->i_h=(int*) realloc(data->i_h,data->NDmax*sizeof(int));
      }
      n++;

      linebuffer=line;
      sscanf(linebuffer,"%lf%n",&E,&ibuffer);
      linebuffer+=ibuffer;
      data->D_h[data->ND*data->Ndim]=E;
      for (j=0; j<data->NF; j++) {
        sscanf(linebuffer,"%lf%n",&E,&ibuffer);
        linebuffer+=ibuffer;
        data->D_h[data->ND*data->Ndim+data->NL+1+j]=E;
      }
      // data->D_h[data->ND*data->Ndim]=E;

      fgets(line,MAXLENGTH,fpQ);
      linebuffer=line;
      for (j=0; j<data->Nblocks; j++) {
        sscanf(linebuffer,"%lf%n",&q,&ibuffer);
        linebuffer+=ibuffer;
        data->D_h[data->ND*data->Ndim+1+j]=q;
      }

      data->i_h[data->ND]=i;

      iB0=(int) floor(E/data->B[0].dx);
      // iB1=(int) floor(q/data->B[1].dx);
      if (iB0<B0_MIN) {
        B0_MIN=iB0;
      }
      if (iB0>B0_MAX) {
        B0_MAX=iB0;
      }
      // if (iB1<B1_MIN) {
      //   B1_MIN=iB1;
      // }
      // if (iB1>B1_MAX) {
      //   B1_MAX=iB1;
      // }
      
      data->ND++;
    }

    data->n_h[i]=n;

    fclose(fpE);
    fclose(fpQ);
  }

  data->B[0].min=B0_MIN*data->B[0].dx;
  data->B[0].max=(B0_MAX+1)*data->B[0].dx;
  data->B[0].N=(B0_MAX-B0_MIN)+1;

  data->B2d[0].min=data->B[0].min;
  data->B2d[0].max=data->B[0].max;
  data->B2d[0].N=data->B[0].N;

  data->B[1].min=0;
  data->B[1].max=1;
  data->B[1].N=400;
  B_N=data->B[1].N;

  data->B2d[1].min=0;
  data->B2d[1].max=1;
  data->B2d[1].N=20;
  data->B2d[2].min=0;
  data->B2d[2].max=1;
  data->B2d[2].N=20;
  if (data->B2d[1].N*data->B2d[2].N>B_N) {
    B_N=data->B2d[1].N*data->B2d[2].N;
  }

  hipMemcpy(data->n_d,data->n_h,data->NF*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc(&(data->D_d),data->NDmax*data->Ndim*sizeof(double));
  hipMemcpy(data->D_d,data->D_h,data->NDmax*data->Ndim*sizeof(double),hipMemcpyHostToDevice);

  hipMalloc(&(data->i_d),data->NDmax*sizeof(int));
  hipMemcpy(data->i_d,data->i_h,data->NDmax*sizeof(int),hipMemcpyHostToDevice);

  hipMalloc(&(data->lnw_d),data->NF*sizeof(double));
  hipMemcpy(data->lnw_d,data->lnw_h,data->NF*sizeof(double),hipMemcpyHostToDevice);

  data->lnDenom_h=(double*) malloc(data->NDmax*sizeof(double));
  hipMalloc(&(data->lnDenom_d),data->NDmax*sizeof(double));

  /*
  V=[None]*2

  for iB in range(0,2):
    V[iB]=[None]*B[iB].N
    for i in range(0,B[iB].N):
      V[iB][i]=(B[iB].max-B[iB].min)/B[iB].N*i+B[iB].min
  */

  data->f_h=(double*) malloc(data->NF*sizeof(double));
  hipMalloc(&(data->f_d),data->NF*sizeof(double));
  for (i=0; i<data->NF; i++) {
    data->f_h[i]=0.0;
  }
  hipMemcpy(data->f_d,data->f_h,data->NF*sizeof(double),hipMemcpyHostToDevice);
  data->invf_h=(double*) malloc(data->NF*sizeof(double));
  hipMalloc(&(data->invf_d),data->NF*sizeof(double));

  fprintf(stderr,"Warning, DOS allocation is not sparse, requesting %d doubles\n",data->B[0].N*B_N);
  data->lnZ_h=(double*) malloc(B_N*sizeof(double));
  hipMalloc(&(data->lnZ_d),B_N*sizeof(double));

  iN=0;
  for (s1=0; s1<data->Nsites; s1++) {
    for (s2=s1; s2<data->Nsites; s2++) {
      if (s1==s2) {
        if (data->Nsubs[s1]==2) {
          iN+=data->Nsubs[s1]+data->Nsubs[s1]*(data->Nsubs[s1]-1)/2;
        } else {
          iN+=data->Nsubs[s1]+2*data->Nsubs[s1]*(data->Nsubs[s1]-1)/2;
        }
      } else if (data->msprof) {
        iN+=data->Nsubs[s1]*data->Nsubs[s2];
      }
    }
  }
  data->iN=iN;

  jN=0;
  // data->jNij=(int*) malloc((data->Nsites*(data->Nsites+1))/2+1,sizeof(int));
  // i=0;
  for (s1=0; s1<data->Nsites; s1++) {
    for (s2=s1; s2<data->Nsites; s2++) {
      // data->jNij[i]=jN;
      if (s1==s2) {
        jN+=data->Nsubs[s1]+5*data->Nsubs[s1]*(data->Nsubs[s1]-1)/2;
      } else if (data->ms) {
        jN+=5*data->Nsubs[s1]*data->Nsubs[s2];
      }
      // i++;
    }
  }
  // data->jNij[i]=jN;
  data->jN=jN;

  data->dlnZ_hN=(double**) malloc(jN*sizeof(double*));
  for (j=0; j<jN; j++) {
    data->dlnZ_hN[j]=(double*) malloc(B_N*sizeof(double));
  }
  hipMalloc(&(data->dlnZ_d),B_N*sizeof(double));
  hipMalloc(&(data->dlnZ_dN),jN*B_N*sizeof(double));
  data->Gimp_h=(double*) malloc(B_N*sizeof(double));
  hipMalloc(&(data->Gimp_d),B_N*sizeof(double));

  data->C_h=(double*) malloc(jN*sizeof(double));
  hipMalloc(&(data->C_d),jN*sizeof(double));
  data->CV_h=(double*) malloc(jN*sizeof(double));
  hipMalloc(&(data->CV_d),jN*sizeof(double));
  data->CC_h=(double*) malloc(jN*jN*sizeof(double));
  hipMalloc(&(data->CC_d),jN*jN*sizeof(double));
  
  // data->one_h=(double*) malloc(B_N*sizeof(double));
  // data->E_h=(double*) malloc(B_N*sizeof(double));
  // data->E2_h=(double*) malloc(B_N*sizeof(double));

  return data;
}

__global__ void resetlogdata(double *d,int N)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    d[i]=-INFINITY;
  }
}

__global__ void resetdata(double *d,int N)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    d[i]=0.0;
  }
}

__global__ void sumdenom(struct_data data)
{
  int t=blockIdx.x*blockDim.x+threadIdx.x;
  int i;
  double lnwn;
  double E;
  double lnDenom=-INFINITY;

  if (t<data.ND) {
    for (i=0; i<data.NF; i++) {
      E=data.D_d[t*data.Ndim+data.NL+1+i];
      lnwn=data.lnw_d[i]+log((double) data.n_d[i]);
      lnDenom=logadd(lnDenom,lnwn+data.f_d[i]-data.beta_d[i]*E);
    }
    data.lnDenom_d[t]=lnDenom;
  }
}

__global__
void getf(struct_data data)
{
  int t,tmin,tmax;
  int i;
  double lnw;
  double beta;
  double E;
  __shared__ double invf[BLOCK];

  tmin=(data.ND*threadIdx.x)/blockDim.x;
  tmax=(data.ND*(threadIdx.x+1))/blockDim.x;

  i=blockIdx.x;

  beta=data.beta_d[i];

  invf[threadIdx.x]=-INFINITY;
  for (t=tmin; t<tmax; t++) {
    // E=data.D_d[t][0];
    lnw=data.lnw_d[data.i_d[t]];
    E=data.D_d[t*data.Ndim+data.NL+1+i];
    invf[threadIdx.x]=logadd(invf[threadIdx.x],lnw-beta*E-data.lnDenom_d[t]);
  }

  __syncthreads();

  for (t=1; t<blockDim.x; t*=2) {
    if ((threadIdx.x % (2*t)) == 0) {
      invf[threadIdx.x]=logadd(invf[threadIdx.x],invf[threadIdx.x+t]);
    }
    __syncthreads();
  }

  if (threadIdx.x==0) {
    data.invf_d[i]=invf[0];
    data.f_d[i]=-invf[0];
  }
}

__global__ void normf(double *f,double f_avg,int N)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    f[i]-=f_avg;
  }
}

void iteratedata(struct_data *data)
{
  int escape_flag=0;
  int itt;
  int mitt=1000;
  int i;
  double f_sum;
  FILE *fp;

  for (itt=0; itt<mitt; itt++) {
    escape_flag=1;
    // resetlogdata <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data->lnDenom_d,data->ND);
    sumdenom <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0]);
    // resetlogdata <<< (data->NF+BLOCK-1)/BLOCK, BLOCK >>> (data->invf_d,data->NF);
    getf <<< data->NF, BLOCK >>> (data[0]);

    hipMemcpy(data->invf_h,data->invf_d,data->NF*sizeof(double),hipMemcpyDeviceToHost);
    f_sum=0;
    for (i=0; i<data->NF; i++) {
      fprintf(stdout," %f",data->f_h[i]);
      // fprintf(stderr,"%d %d %f %f\n",itt,i,data->f_h[i],-data->invf_h[i]);
      if (data->f_h[i]+data->invf_h[i]<-0.00005 || data->f_h[i]+data->invf_h[i]>0.00005){
        escape_flag=0;
      }
      data->f_h[i]=-data->invf_h[i];
      f_sum+=data->f_h[i];
    }
    fprintf(stdout,"\n");
    f_sum/=data->NF;
    normf <<< (data->NF+BLOCK-1)/BLOCK,BLOCK >>> (data->f_d,f_sum,data->NF);
    if (escape_flag==1) {
      break;
    }
  }

  fp=fopen("f.dat","w");
  for (i=0; i<data->NF; i++) {
    fprintf(fp," %12.5f",data->f_h[i]);
  }
  fclose(fp);
}

__global__ void bin1(struct_data data,int i1)
{
  double q1;
  int t;
  int iB1;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  if (t<data.ND) {
    q1=data.D_d[t*data.Ndim+1+i1];
    iB1=(int) floor((q1-data.B[1].min)/data.B[1].dx);
    data.D_d[t*data.Ndim+1+data.NL+data.Nsim]=iB1;
  }
}

__global__ void bin12(struct_data data,int i1,int i2)
{
  double q1,q2,q;
  int t;
  int iB12;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  if (t<data.ND) {
    q1=data.D_d[t*data.Ndim+1+i1];
    q2=data.D_d[t*data.Ndim+1+i2];
    if (q1+q2>0.8) {
      q=q1/(q1+q2);
      iB12=(int) floor((q-data.B[1].min)/data.B[1].dx);
    } else {
      iB12=-1;
    }
    data.D_d[t*data.Ndim+1+data.NL+data.Nsim]=iB12;
  }
}

__global__ void bin2(struct_data data,int i1,int i2)
{
  double q1,q2;
  int t;
  int iB1,iB2;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  if (t<data.ND) {
    q1=data.D_d[t*data.Ndim+1+i1];
    q2=data.D_d[t*data.Ndim+1+i2];
    iB1=(int) floor((q1-data.B2d[1].min)/data.B2d[1].dx);
    iB2=(int) floor((q2-data.B2d[2].min)/data.B2d[2].dx);
    data.D_d[t*data.Ndim+1+data.NL+data.Nsim]=iB1*data.B2d[1].N+iB2;
  }
}

double bin_all(struct_data *data,int *ptype,int i)
{
  double wnorm;
  int s1,s2;
  int i1,i2;
  int B_N;
  char fnm[MAXLENGTH];
  FILE *fp;

  for (s1=0; s1<data->Nsites; s1++) {
    for (s2=s1; s2<data->Nsites; s2++) {

      if (s1==s2) {

        for (i1=data->block0[s1]; i1<data->block0[s1+1]; i1++) {
          if (i==0) {
            fprintf(stderr,"1D Profile %d\n",i1);
            wnorm=1.0;
            bin1 <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],i1);
            sprintf(fnm,"../ALF/G_imp/G1_%d.dat",data->Nsubs[s1]);
            B_N=data->B[1].N;
            ptype[0]=0;
          }
          i--;
        }

        for (i1=data->block0[s1]; i1<data->block0[s1+1]; i1++) {
          for (i2=i1+1; i2<data->block0[s1+1]; i2++) {
            if (i==0) {
              fprintf(stderr,"1D Profile %d,%d\n",i1,i2);
              wnorm=1.0/((data->Nsubs[s1]-1)/2.0);
              bin12 <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],i1,i2);
              sprintf(fnm,"../ALF/G_imp/G12_%d.dat",data->Nsubs[s1]);
              B_N=data->B2d[1].N*data->B2d[2].N;
              ptype[0]=1;
            }
            i--;
          }
        }

        if (data->Nsubs[s1]>2) {
          for (i1=data->block0[s1]; i1<data->block0[s1+1]; i1++) {
            for (i2=i1+1; i2<data->block0[s1+1]; i2++) {
              if (i==0) {
                fprintf(stderr,"2D Profile %d,%d\n",i1,i2);
                wnorm=1.0/((data->Nsubs[s1]-1)/2.0);
                bin2 <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],i1,i2);
                sprintf(fnm,"../ALF/G_imp/G2_%d.dat",data->Nsubs[s1]);
                B_N=data->B2d[1].N*data->B2d[2].N;
                ptype[0]=2;
              }
              i--;
            }
          }
        }

      } else if (data->msprof) { // Site-site interaction

        for (i1=data->block0[s1]; i1<data->block0[s1+1]; i1++) {
          for (i2=data->block0[s2]; i2<data->block0[s2+1]; i2++) {
            if (i==0) {
              fprintf(stderr,"2D SS Profile %d,%d\n",i1,i2);
              wnorm=1.0/(data->Nsubs[s1]*data->Nsubs[s2]);
              bin2 <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],i1,i2);
              sprintf(fnm,"../ALF/G_imp/G1_%d_%d.dat",data->Nsubs[s1],data->Nsubs[s2]);
              B_N=data->B2d[1].N*data->B2d[2].N;
              ptype[0]=3;
            }
            i--;
          }
        }

      }
    }
  }

  fp=fopen(fnm,"r");
  if (fp==NULL) {
    fprintf(stderr,"Error, %s does not exist\n",fnm);
    exit(1);
  }
  for (i=0; i<B_N; i++) {
    fscanf(fp,"%lf",&data->Gimp_h[i]);
  }
  fclose(fp);

  hipMemcpy(data->Gimp_d,data->Gimp_h,B_N*sizeof(double),hipMemcpyHostToDevice);

  return wnorm;
}

// slope
__global__ void reactioncoord_phi(struct_data data,int i1)
{
  double q1;
  int t;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  if (t<data.ND) {
    q1=data.D_d[t*data.Ndim+1+i1];
    data.D_d[t*data.Ndim+1+data.NL+data.Nsim+1]=q1;
  }
}

// quadratic
__global__ void reactioncoord_psi(struct_data data,int i1,int i2)
{
  double q1,q2;
  int t;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  if (t<data.ND) {
    q1=data.D_d[t*data.Ndim+1+i1];
    q2=data.D_d[t*data.Ndim+1+i2];
    data.D_d[t*data.Ndim+1+data.NL+data.Nsim+1]=q1*q2;
  }
}

// omega - sharp endpoint
__global__ void reactioncoord_omega(struct_data data,int i1,int i2)
{
  double q1,q2;
  int t;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  if (t<data.ND) {
    q1=data.D_d[t*data.Ndim+1+i1];
    q2=data.D_d[t*data.Ndim+1+i2];
    data.D_d[t*data.Ndim+1+data.NL+data.Nsim+1]=q2*(1-1/(q1/0.017+1));
  }
}

// slope
__global__ void reactioncoord_chi(struct_data data,int i1,int i2)
{
  double q1,q2;
  int t;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  if (t<data.ND) {
    q1=data.D_d[t*data.Ndim+1+i1];
    q2=data.D_d[t*data.Ndim+1+i2];
    data.D_d[t*data.Ndim+1+data.NL+data.Nsim+1]=q2*(1-exp(-q1/0.18));
  }
}

void reactioncoord_all(struct_data *data,int i)
{
  int s1,s2;
  int j1,j2;

  for (s1=0; s1<data->Nsites; s1++) {
    for (s2=s1; s2<data->Nsites; s2++) {

      if (s1==s2) {

        for (j1=data->block0[s1]; j1<data->block0[s1+1]; j1++) {
          if (i==0) {
            reactioncoord_phi <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j1);
          }
          i--;
        }

        for (j1=data->block0[s1]; j1<data->block0[s1+1]; j1++) {
          for (j2=j1+1; j2<data->block0[s1+1]; j2++) {
            if (i==0) {
              reactioncoord_psi <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j1,j2);
            }
            i--;
          }
        }

        for (j1=data->block0[s1]; j1<data->block0[s1+1]; j1++) {
          for (j2=data->block0[s1]; j2<data->block0[s1+1]; j2++) {
            if (j1 != j2) {
              if (i==0) {
                reactioncoord_chi <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j1,j2);
              }
              i--;
            }
          }
        }

        for (j1=data->block0[s1]; j1<data->block0[s1+1]; j1++) {
          for (j2=data->block0[s1]; j2<data->block0[s1+1]; j2++) {
            if (j1 != j2) {
              if (i==0) {
                reactioncoord_omega <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j1,j2);
              }
              i--;
            }
          }
        }

      } else if (data->ms) { // Different sites

        for (j1=data->block0[s1]; j1<data->block0[s1+1]; j1++) {
          for (j2=data->block0[s2]; j2<data->block0[s2+1]; j2++) {
            if (i==0) {
              reactioncoord_psi <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j1,j2);
            }
            i--;
          }
        }

        for (j1=data->block0[s1]; j1<data->block0[s1+1]; j1++) {
          for (j2=data->block0[s2]; j2<data->block0[s2+1]; j2++) {
            if (i==0) {
              reactioncoord_chi <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j1,j2);
            }
            i--;
            if (i==0) {
              reactioncoord_chi <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j2,j1);
            }
            i--;
          }
        }

        for (j1=data->block0[s1]; j1<data->block0[s1+1]; j1++) {
          for (j2=data->block0[s2]; j2<data->block0[s2+1]; j2++) {
            if (i==0) {
              reactioncoord_omega <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j1,j2);
            }
            i--;
            if (i==0) {
              reactioncoord_omega <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0],j2,j1);
            }
            i--;
          }
        }

      }
    }
  }
}

__global__ void get_lnZ(struct_data data,double beta)
{
  double E;
  double lnw;
  int t,i;
  double *p_lnZ;
  __shared__ double loc_lnZ[400];
  int iB;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadIdx.x; i<400; i+=blockDim.x) {
    loc_lnZ[i]=-INFINITY;
  }

  __syncthreads();

  for (i=SBLOCK*t; i<SBLOCK*(t+1); i++) {
    if (i<data.ND) {
      E=data.D_d[i*data.Ndim+0];
      iB=(int) data.D_d[i*data.Ndim+1+data.NL+data.Nsim];
      if (iB>=0) {
        lnw=data.lnw_d[data.i_d[i]];
        p_lnZ=&loc_lnZ[iB];
        atomic_logadd(p_lnZ,lnw-data.lnDenom_d[i]-beta*E);
      }
    }
  }

  __syncthreads();

  for (i=threadIdx.x; i<400; i+=blockDim.x) {
    if (loc_lnZ[i]>-INFINITY) {
      p_lnZ=&data.lnZ_d[i];
      atomic_logadd(p_lnZ,loc_lnZ[i]);
    }
  }
}

__global__ void get_dlnZ(struct_data data,int j1,double beta)
{
  double E,q;
  double lnw;
  int t,i;
  double *p_dlnZ;
  __shared__ double loc_dlnZ[400];
  int iB;

  t=blockIdx.x*blockDim.x+threadIdx.x;

  for (i=threadIdx.x; i<400; i+=blockDim.x) {
    loc_dlnZ[i]=-INFINITY;
  }

  __syncthreads();

  for (i=SBLOCK*t; i<SBLOCK*(t+1); i++) {
    if (i<data.ND) {
      E=data.D_d[i*data.Ndim+0];
      iB=(int) data.D_d[i*data.Ndim+1+data.NL+data.Nsim];
      q=data.D_d[i*data.Ndim+1+data.NL+data.Nsim+1];
      if (iB>=0) {
        lnw=data.lnw_d[data.i_d[i]];
        p_dlnZ=&loc_dlnZ[iB];
        atomic_logadd(p_dlnZ,lnw-data.lnDenom_d[i]-beta*E+log(q));
      }
    }
  }

  __syncthreads();

  for (i=threadIdx.x; i<400; i+=blockDim.x) {
    if (loc_dlnZ[i]>-INFINITY) {
      p_dlnZ=&data.dlnZ_dN[400*j1+i];
      atomic_logadd(p_dlnZ,loc_dlnZ[i]);
    }
  }
}

__global__ void get_CC(struct_data data,int i,double beta,double wnorm,int ptype)
{
  int j1=blockIdx.x;
  int j2=blockIdx.y;
  int k;
  double weight;
  double lnZ, dlnZ1, dlnZ2;
  double myCC;
  __shared__ double loc_CC[100];

  myCC=0;

  for (k=threadIdx.x;k<400;k+=100) {
    weight=wnorm;
    if ((ptype==0 || ptype==3) && k==400-1) {
      weight*=100.0;
    }
    lnZ=data.lnZ_d[k];
    if (isfinite(lnZ)) {
      dlnZ1=data.dlnZ_dN[j1*400+k];
      dlnZ2=data.dlnZ_dN[j2*400+k];
      myCC+=weight*exp(dlnZ1-lnZ)*exp(dlnZ2-lnZ);
    }
  }

  loc_CC[threadIdx.x]=myCC;

  __syncthreads();

  for (k=1; k<100; k*=2) {
    if (threadIdx.x%(2*k) == 0) {
      if (threadIdx.x+k < 100) {
        loc_CC[threadIdx.x]+=loc_CC[threadIdx.x+k];
      }
    }
    __syncthreads();
  }

  if (threadIdx.x==0) {
    // jN=data->NL+5*data->NL*(data->NL-1)/2;
    // jN=gridDim.x;
    data.CC_d[gridDim.x*j1+j2]=loc_CC[0];
  }
}

void getfofq(struct_data *data,double beta)
{
  int B_N;
  int i,iN;
  int j1,j2,jN;
  int k;
  double *C, *V;
  double wnorm;
  int ptype; // profile type, affects bin weight
  double weight;
  char fnm[MAXLENGTH];
  FILE *fpC,*fpV,*fp;

  B_N=data->B[1].N;
  if (data->B2d[1].N*data->B2d[2].N>B_N) {
    B_N=data->B2d[1].N*data->B2d[2].N;
  }

  iN=data->iN;
  jN=data->jN;

  C=(double*) malloc((jN+iN)*(jN+iN)*sizeof(double));
  V=(double*) malloc((jN+iN)*sizeof(double));
  for (j1=0;j1<(jN+iN);j1++) {
    for (j2=0;j2<(jN+iN);j2++) {
      C[j1*(jN+iN)+j2]=0;
    }
    V[j1]=0;
  }

  sumdenom <<< (data->ND+BLOCK-1)/BLOCK, BLOCK >>> (data[0]);

  for (i=0; i<iN; i++) {
    wnorm=bin_all(data,&ptype,i);

    resetlogdata <<< (B_N+BLOCK-1)/BLOCK, BLOCK >>> (data->lnZ_d,B_N);
    get_lnZ <<< (data->ND+(100*SBLOCK)-1)/(100*SBLOCK), 100 >>> (data[0],data->beta_t);
    hipMemcpy(data->lnZ_h,data->lnZ_d,B_N*sizeof(double),hipMemcpyDeviceToHost);

    sprintf(fnm,"G%d.dat",i+1);
    fp=fopen(fnm,"w");
    for (k=0; k<B_N; k++) {
      fprintf(fp,"%g\n",(-data->lnZ_h[k]-data->Gimp_h[k])/data->beta_t);
    }
    fclose(fp);

    for (k=0;k<B_N;k++) {
      weight=wnorm;
      if ((ptype==0 || ptype==3) && k==400-1) {
        weight*=100.0;
      }
      if (isfinite(data->lnZ_h[k])) {
        V[jN+i]+=weight*(-data->lnZ_h[k]-data->Gimp_h[k])/data->beta_t;
        C[(jN+i)*(jN+iN)+jN+i]+=weight;
      }
      // if (C[(jN+i)*(jN+iN)+jN+i]==0) {
      //   C[(jN+i)*(jN+iN)+jN+i]=1.0;
      // }
    }

    for (j1=0; j1<jN; j1++) {
      reactioncoord_all(data,j1);
      resetlogdata <<< (B_N+BLOCK-1)/BLOCK, BLOCK >>> (&(data->dlnZ_dN[B_N*j1]),B_N);
      get_dlnZ <<< (data->ND+(100*SBLOCK)-1)/(100*SBLOCK), 100 >>> (data[0],j1,data->beta_t);
      hipMemcpy(data->dlnZ_hN[j1],&(data->dlnZ_dN[B_N*j1]),B_N*sizeof(double),hipMemcpyDeviceToHost);

      /*
      sprintf(fnm,"dG%d_d%d.dat",i+1,j1+1);
      fp=fopen(fnm,"w");
      for (k=0; k<B_N; k++) {
        fprintf(fp,"%g\n",exp(data->dlnZ_hN[j1][k]-data->lnZ_h[k]));
      }
      fclose(fp);
      */
    }

    get_CC <<< make_uint3(jN,jN,1), 100 >>> (data[0],i,data->beta_t,wnorm,ptype);
    hipMemcpy(data->CC_h,data->CC_d,jN*jN*sizeof(double),hipMemcpyDeviceToHost);

    for (j1=0; j1<jN; j1++) {
      for (k=0;k<B_N;k++) {
        weight=wnorm;
        if ((ptype==0 || ptype==3) && k==400-1) {
          weight*=100.0;
        }
        if (isfinite(data->lnZ_h[k])) {
          if (isfinite(data->Gimp_h[k])==0) {
            fprintf(stderr,"Fatal error, implicit constraint entropy is undefined at bin %d\n",k);
            exit(1);
          }
          V[j1]+=weight*exp(data->dlnZ_hN[j1][k]-data->lnZ_h[k])*(-data->lnZ_h[k]-data->Gimp_h[k])/data->beta_t;
          C[j1*(jN+iN)+jN+i]+=weight*exp(data->dlnZ_hN[j1][k]-data->lnZ_h[k]);
          C[(jN+i)*(jN+iN)+j1]+=weight*exp(data->dlnZ_hN[j1][k]-data->lnZ_h[k]);
        }
      }

      for (j2=0; j2<jN; j2++) {
        /*
        for (k=0;k<B_N;k++) {
          if (wnorm==1.0 && k==B_N-1) {
            weight=100.0;
          } else {
            weight=wnorm;
          }
          if (isfinite(data->lnZ_h[k])) {
            C[j1*(jN+iN)+j2]+=weight*exp(data->dlnZ_hN[j1][k]-data->lnZ_h[k])*exp(data->dlnZ_hN[j2][k]-data->lnZ_h[k]);
          }
        }
        */
        C[j1*(jN+iN)+j2]+=data->CC_h[j1*jN+j2];
      }
    }
  }

  fpC=fopen("C.dat","w");
  fpV=fopen("V.dat","w");
  for (j1=0; j1<(jN+iN); j1++) {
    for (j2=0; j2<(jN+iN); j2++) {
      fprintf(fpC," %f",C[j1*(jN+iN)+j2]);
    }
    fprintf(fpC,"\n");
    fprintf(fpV," %f\n",V[j1]);
  }
  fclose(fpC);
  fclose(fpV);
}

int main(int argc, char *argv[])
{
  struct_data *data;
  
  data=readdata(argc,argv);

  iteratedata(data);

  getfofq(data,data->beta_t);
}

