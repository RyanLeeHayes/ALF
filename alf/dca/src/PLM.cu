#include "hip/hip_runtime.h"
// Written by Ryan Hayes 2017-06-20
// plmDCA algorithm from R470 - DOI: 10.1016/j.jcp.2014.07.024
// Quasi newton equations from https://www.rose-hulman.edu/~bryan/lottamath/quasinewton.pdf

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <time.h>
#include <unistd.h>

//   ID=omp_get_thread_num();
//   NID=omp_get_max_threads();

#include "PLM.h"

#define MAXLENGTH 1024
#define PLMGPU

struct_plmd* setup(int argc, char *argv[])
{
  struct_plmd *plmd;
  int h,i,j,k,l;
  FILE *fp;
  char line[MAXLENGTH];
  char *linebuf;

  if (argc<5) {
    fprintf(stderr,"Error: not enough input arguments\n");
    exit(1);
  }

  plmd=(struct_plmd*) malloc(sizeof(struct_plmd));

  fp=fopen("../prep/nsubs","r");
  if (fp==NULL) {
    fprintf(stderr,"Error, ../prep/nsubs does not exist\n");
    exit(1);
  }
  plmd->nsites=0;
  while (fscanf(fp,"%d",&i)==1) {
    plmd->nsites++;
  }
  fclose(fp);

  fp=fopen("../prep/nsubs","r");
  i=0;
  plmd->nsubs=(int*) calloc(plmd->nsites,sizeof(int));
  plmd->nblocks=0;
  plmd->nsubsmax=0;
  for(i=0; i<plmd->nsites; i++) {
    fscanf(fp,"%d",&(plmd->nsubs[i]));
    plmd->nsubs[i]++;
    plmd->nblocks+=plmd->nsubs[i];
    plmd->nsubsmax=(plmd->nsubs[i]>plmd->nsubsmax?plmd->nsubs[i]:plmd->nsubsmax);
  }
  fclose(fp);

  plmd->block0=(int*) calloc(plmd->nsites+1,sizeof(int));
  plmd->block2site=(int*) calloc(plmd->nblocks,sizeof(int));
  k=0;
  for(i=0; i<plmd->nsites; i++) {
    plmd->block0[i]=k;
    for(j=0; j<plmd->nsubs[i]; j++) {
      plmd->block2site[k]=i;
      k++;
    }
  }
  plmd->block0[i]=k;

#ifdef PLMGPU
  hipMalloc(&plmd->block0_d,(plmd->nsites+1)*sizeof(int));
  hipMemcpy(plmd->block0_d,plmd->block0,(plmd->nsites+1)*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc(&plmd->nsubs_d,plmd->nsites*sizeof(int));
  hipMemcpy(plmd->nsubs_d,plmd->nsubs,plmd->nsites*sizeof(int),hipMemcpyHostToDevice);
#endif

  plmd->h=(real*) calloc(plmd->nblocks,sizeof(real));
  plmd->J=(real*) calloc(plmd->nblocks*plmd->nblocks,sizeof(real));

  plmd->dhds=(real*) calloc(plmd->nblocks,sizeof(real));
  plmd->dJds=(real*) calloc(plmd->nblocks*plmd->nblocks,sizeof(real));

  plmd->dLdh=(reala*) calloc(plmd->nblocks,sizeof(reala));
  plmd->dLdJ=(reala*) calloc(plmd->nblocks*plmd->nblocks,sizeof(reala));

#ifdef PLMGPU
  hipMalloc(&plmd->h_d,plmd->nblocks*sizeof(real));
  hipMalloc(&plmd->J_d,plmd->nblocks*plmd->nblocks*sizeof(real));

  hipMalloc(&plmd->dhds_d,plmd->nblocks*sizeof(real));
  hipMalloc(&plmd->dJds_d,plmd->nblocks*plmd->nblocks*sizeof(real));

  hipMalloc(&plmd->dLdh_d,plmd->nblocks*sizeof(reala));
  hipMalloc(&plmd->dLdJ_d,plmd->nblocks*plmd->nblocks*sizeof(reala));
#endif

  plmd->B=0;
  for (h=0; h<argc-3; h++) {
  fp=fopen(argv[h+3],"r");
  for (; fgets(line,MAXLENGTH,fp) != NULL; plmd->B++) {
    ;
  }
  fclose(fp);
  }
  fprintf(stderr,"%d frames\n",plmd->B); // DEBUG

  plmd->BiBmax=(1<<20);
  if (plmd->B>plmd->BiBmax) { // Too much data, have to save to temporary files
    plmd->Seq_fp=tmpfile();
    plmd->BiB=plmd->BiBmax;
  } else { // All fits in memory
    plmd->BiB=plmd->B;
  }

  plmd->Seq=(int*) calloc(plmd->BiB*plmd->nsites,sizeof(int));
  i=0;
  for (h=0; h<argc-3; h++) {
  fp=fopen(argv[h+3],"r");
  for (; fgets(line,MAXLENGTH,fp) != NULL; i++) {
    linebuf=line;
    k=0;
    for (j=0;j<plmd->nsites;j++) {
      sscanf(linebuf,"%d%n",&(plmd->Seq[i*plmd->nsites+j]),&l);
      linebuf+=l;
      plmd->Seq[i*plmd->nsites+j]+=k;
      k+=plmd->nsubs[j];
    }
    if (plmd->B>plmd->BiBmax && i>=plmd->BiBmax) {
      fwrite(plmd->Seq,sizeof(int),i*plmd->nsites,plmd->Seq_fp);
      i=0;
    }
  }
  fclose(fp);
  }
  if (plmd->B>plmd->BiBmax) {
    fwrite(plmd->Seq,sizeof(int),i*plmd->nsites,plmd->Seq_fp);
  }

#ifdef PLMGPU
  hipMalloc(&plmd->Seq_d,plmd->BiB*plmd->nsites*sizeof(int));
  hipMemcpy(plmd->Seq_d,plmd->Seq,plmd->BiB*plmd->nsites*sizeof(int),hipMemcpyHostToDevice);
#endif

  plmd->heff=(real*) calloc(plmd->BiB*plmd->nblocks,sizeof(real));
  plmd->dheffds=(real*) calloc(plmd->BiB*plmd->nblocks,sizeof(real));

#ifdef PLMGPU
  hipMalloc(&plmd->heff_d,plmd->BiB*plmd->nblocks*sizeof(real));
  hipMalloc(&plmd->dheffds_d,plmd->BiB*plmd->nblocks*sizeof(real));
#endif

  plmd->kh=1e-6;
  plmd->kJ=1e-6;

  plmd->L=(reala*) calloc(1,sizeof(reala));
  plmd->dLds=(reala*) calloc(1,sizeof(reala));

#ifdef PLMGPU
  hipMalloc(&plmd->L_d,sizeof(reala));
  hipMalloc(&plmd->dLds_d,sizeof(reala));
#endif


  k=plmd->nblocks;
  for (i=0; i<plmd->nblocks; i++) {
    for (j=plmd->block0[plmd->block2site[i]+1]; j<plmd->nblocks; j++) {
      k++;
    }
  }
  plmd->Jend=k;

  plmd->x=(real*) calloc(plmd->Jend,sizeof(real));
  plmd->dLdx=(reala*) calloc(plmd->Jend,sizeof(reala));
  // plmd->Hinv=(real*) calloc(plmd->Jend*plmd->Jend,sizeof(real));
  plmd->Nmemax=200;
  plmd->Nmem=0;
  plmd->d_x=(real*) calloc(plmd->Jend*plmd->Nmemax,sizeof(real));
  plmd->d_dLdx=(real*) calloc(plmd->Jend*plmd->Nmemax,sizeof(real));
  plmd->rho=(real*) calloc(plmd->Nmemax,sizeof(real));
  plmd->alpha=(real*) calloc(plmd->Jend*plmd->Nmemax,sizeof(real));
  plmd->beta=(real*) calloc(plmd->Jend*plmd->Nmemax,sizeof(real));

  plmd->hi=(real*) calloc(plmd->Jend,sizeof(real));
  plmd->x0=(real*) calloc(plmd->Jend,sizeof(real));
  plmd->dLdx0=(reala*) calloc(plmd->Jend,sizeof(reala));

  // for (i=0; i<plmd->Jend*plmd->Jend; i++) {
  //   plmd->Hinv[i]=0;
  // }
  // for (i=0; i<plmd->Jend; i++) {
  //   plmd->x[i]=0;
  //   plmd->Hinv[i*(plmd->Jend+1)]=1;
  // }

  /*{
    hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType=hipResourceTypeLinear;
    resDesc.res.linear.devPtr=plmd->h_d;
    resDesc.res.linear.desc=hipCreateChannelDesc<real>();
    resDesc.res.linear.sizeInBytes=plmd->nblocks*sizeof(real);
    hipTextureDesc texDesc;
    memset(&texDesc,0,sizeof(texDesc));
    texDesc.readMode=hipReadModeElementType;
    hipCreateTextureObject(&plmd->h_tex,&resDesc,&texDesc,NULL);
  }
  {
    hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType=hipResourceTypeLinear;
    resDesc.res.linear.devPtr=plmd->J_d;
    resDesc.res.linear.desc=hipCreateChannelDesc<real>();
    resDesc.res.linear.sizeInBytes=plmd->nblocks*plmd->nblocks*sizeof(real);
    hipTextureDesc texDesc;
    memset(&texDesc,0,sizeof(texDesc));
    texDesc.readMode=hipReadModeElementType;
    hipCreateTextureObject(&plmd->J_tex,&resDesc,&texDesc,NULL);
  }
  {
    hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType=hipResourceTypeLinear;
    resDesc.res.linear.devPtr=plmd->dhds_d;
    resDesc.res.linear.desc=hipCreateChannelDesc<real>();
    resDesc.res.linear.sizeInBytes=plmd->nblocks*sizeof(real);
    hipTextureDesc texDesc;
    memset(&texDesc,0,sizeof(texDesc));
    texDesc.readMode=hipReadModeElementType;
    hipCreateTextureObject(&plmd->dhds_tex,&resDesc,&texDesc,NULL);
  }
  {
    hipResourceDesc resDesc;
    memset(&resDesc,0,sizeof(resDesc));
    resDesc.resType=hipResourceTypeLinear;
    resDesc.res.linear.devPtr=plmd->dJds_d;
    resDesc.res.linear.desc=hipCreateChannelDesc<real>();
    resDesc.res.linear.sizeInBytes=plmd->nblocks*plmd->nblocks*sizeof(real);
    hipTextureDesc texDesc;
    memset(&texDesc,0,sizeof(texDesc));
    texDesc.readMode=hipReadModeElementType;
    hipCreateTextureObject(&plmd->dJds_tex,&resDesc,&texDesc,NULL);
  }*/

  return plmd;
}

void copyout_hJ(struct_plmd *plmd)
{
  int i,j,k;

  k=plmd->nblocks;
  for (i=0; i<plmd->nblocks; i++) {
    plmd->h[i]=plmd->x[i];
    for (j=plmd->block0[plmd->block2site[i]+1]; j<plmd->nblocks; j++) {
      plmd->J[i*plmd->nblocks+j]=plmd->x[k];
      plmd->J[j*plmd->nblocks+i]=plmd->x[k];
      k++;
    }
  }
}

void copyback_hJ(struct_plmd *plmd)
{
  int i,j,k;

  k=plmd->nblocks;
  for (i=0; i<plmd->nblocks; i++) {
    plmd->x[i]=plmd->h[i];
    for (j=plmd->block0[plmd->block2site[i]+1]; j<plmd->nblocks; j++) {
      plmd->x[k]=plmd->J[i*plmd->nblocks+j];
      k++;
    }
  }
}

void copyback_gradient(struct_plmd *plmd)
{
  int i,j,k;

  k=plmd->nblocks;
  for (i=0; i<plmd->nblocks; i++) {
    plmd->dLdx[i]=plmd->dLdh[i];
    for (j=plmd->block0[plmd->block2site[i]+1]; j<plmd->nblocks; j++) {
      plmd->dLdx[k]=(plmd->dLdJ[i*plmd->nblocks+j]+plmd->dLdJ[j*plmd->nblocks+i]);
      k++;
    }
  }
}

void copyout_gradient(struct_plmd *plmd)
{
  int i,j,k;

  k=plmd->nblocks;
  for (i=0; i<plmd->nblocks; i++) {
    plmd->dhds[i]=plmd->hi[i];
    for (j=plmd->block0[plmd->block2site[i]+1]; j<plmd->nblocks; j++) {
      plmd->dJds[i*plmd->nblocks+j]=plmd->hi[k];
      plmd->dJds[j*plmd->nblocks+i]=plmd->hi[k];
      k++;
    }
  }
}

#ifndef PLMGPU

void regularize_function(struct_plmd plmd)
{
  int i;
  reala L;

#ifdef PROFILE_CPU
  clock_t t1,t2;
  t1=clock();
#endif

  L=0.0;

  for (i=0; i<plmd.nblocks; i++) {
    L+=0.5*plmd.kh*plmd.h[i]*plmd.h[i];
  }

  for (i=0; i<plmd.nblocks*plmd.nblocks; i++) {
    L+=0.5*plmd.kJ*plmd.J[i]*plmd.J[i];
  }

  plmd.L[0]=L;

#ifdef PROFILE_CPU
  t2=clock();
  fprintf(stdout,"regularize_function time=%f\n",(double)(t2-t1)/(double)(CLOCKS_PER_SEC));
#endif
}

void regularize_function_line(real s,struct_plmd plmd)
{
  int i;
  reala L;
  reala dLds;

  L=0.0;
  dLds=0.0;

  for (i=0; i<plmd.nblocks; i++) {
    L+=0.5*plmd.kh*(plmd.h[i]+s*plmd.dhds[i])*(plmd.h[i]+s*plmd.dhds[i]);
    dLds+=plmd.kh*(plmd.h[i]+s*plmd.dhds[i])*plmd.dhds[i];
  }

  for (i=0; i<plmd.nblocks*plmd.nblocks; i++) {
    L+=0.5*plmd.kJ*(plmd.J[i]+s*plmd.dJds[i])*(plmd.J[i]+s*plmd.dJds[i]);
    dLds+=plmd.kJ*(plmd.J[i]+s*plmd.dJds[i])*plmd.dJds[i];
  }

  plmd.L[0]=L;
  plmd.dLds[0]=dLds;
}

void partition_function(int site,int block0,struct_plmd plmd)
{
  int b;
  int i,j;
  real heff;
  int nsubs;

  for (b=0; b<plmd.BiB; b++) {
    nsubs=plmd.nsubs[site];
    for (i=0; i<nsubs; i++) {
      heff=plmd.h[block0+i];
      for (j=0; j<plmd.nsites; j++) {
        heff+=plmd.J[plmd.nblocks*(block0+i) + plmd.Seq[b*plmd.nsites+j]];
      }
      plmd.heff[b*plmd.nblocks+block0+i]=heff;
    }
  }
}

void partition_function_line(int site,int block0,struct_plmd plmd)
{
  int b;
  int i,j;
  real heff;
  int nsubs;

  for (b=0; b<plmd.BiB; b++) {
    nsubs=plmd.nsubs[site];
    for (i=0; i<nsubs; i++) {
      heff=plmd.dhds[block0+i];
      for (j=0; j<plmd.nsites; j++) {
        heff+=plmd.dJds[plmd.nblocks*(block0+i) + plmd.Seq[b*plmd.nsites+j]];
      }
      plmd.dheffds[b*plmd.nblocks+block0+i]=heff;
    }
  }
}

void evaluate_function(int site,int block0,struct_plmd plmd)
{
  int b;
  int i;
  real Z;
  int nsubs;
  reala L;

  L=0.0;

  for (b=0; b<plmd.BiB; b++) {
    nsubs=plmd.nsubs[site];
    L+=-plmd.heff[b*plmd.nblocks+plmd.Seq[b*plmd.nsites+site]];
    Z=0.0;
    for (i=0; i<nsubs; i++) {
      Z+=exp(plmd.heff[b*plmd.nblocks+block0+i]);
    }
    L+=log(Z);
  }

  #pragma omp atomic
  plmd.L[0]+=L/plmd.B;
}

void evaluate_function_line(real s,int site,int block0,struct_plmd plmd)
{
  int b;
  int i;
  real m1,Z,w;
  int nsubs;
  reala dLds, L;

  L=0.0;
  dLds=0.0;

  for (b=0; b<plmd.BiB; b++) {
    nsubs=plmd.nsubs[site];
    L+=-plmd.heff[b*plmd.nblocks+plmd.Seq[b*plmd.nsites+site]];
    L+=-s*plmd.dheffds[b*plmd.nblocks+plmd.Seq[b*plmd.nsites+site]];
    dLds+=-plmd.dheffds[b*plmd.nblocks+plmd.Seq[b*plmd.nsites+site]];
    m1=0.0;
    Z=0.0;
    for (i=0; i<nsubs; i++) {
      w=exp(plmd.heff[b*plmd.nblocks+block0+i]+s*plmd.dheffds[b*plmd.nblocks+block0+i]);
      Z+=w;
      m1+=plmd.dheffds[b*plmd.nblocks+block0+i]*w;
    }
    L+=log(Z);
    dLds+=m1/Z;
  }

  #pragma omp atomic
  plmd.L[0]+=L/plmd.B;
  #pragma omp atomic
  plmd.dLds[0]+=dLds/plmd.B;
}

void regularize_gradient(struct_plmd plmd)
{
  int i;

  for (i=0; i<plmd.nblocks; i++) {
    plmd.dLdh[i]=plmd.kh*plmd.h[i];
  }
  for (i=0; i<plmd.nblocks*plmd.nblocks; i++) {
    plmd.dLdJ[i]=plmd.kJ*plmd.J[i];
  }
}

void evaluate_gradient_h(int site,int block0,struct_plmd plmd)
{
  int b;
  int i;
  int Seq;
  real Z;
  int nsubs;

  for (b=0; b<plmd.BiB; b++) {
    nsubs=plmd.nsubs[site];
    Seq=plmd.Seq[b*plmd.nsites+site]-block0;
    plmd.dLdh[block0+Seq]+=-1.0/plmd.B;
    Z=0;
    for (i=0; i<nsubs; i++) {
      Z+=exp(plmd.heff[b*plmd.nblocks+block0+i]);
    }
    for (i=0; i<nsubs; i++) {
      plmd.dLdh[block0+i]+=exp(plmd.heff[b*plmd.nblocks+block0+i])/Z/plmd.B;
    }
  }
}

void evaluate_gradient_J(int site1,int site2,int block01,int block02,struct_plmd plmd)
{
  int b;
  int i;
  int Seq1,Seq2;
  real Z;
  int nsubs1, nsubs2;

  nsubs1=plmd.nsubs[site1];
  nsubs2=plmd.nsubs[site2];
  for (b=0; b<plmd.BiB; b++) {
    Seq1=plmd.Seq[b*plmd.nsites+site1]-block01;
    Seq2=plmd.Seq[b*plmd.nsites+site2]-block02;
    plmd.dLdJ[plmd.nblocks*(block01+Seq1)+block02+Seq2]+=-1.0/plmd.B;
    Z=0;
    for (i=0; i<nsubs1; i++) {
      Z+=exp(plmd.heff[b*plmd.nblocks+block01+i]);
    }
    for (i=0; i<nsubs1; i++) {
      plmd.dLdJ[plmd.nblocks*(block01+i)+block02+Seq2]+=exp(plmd.heff[b*plmd.nblocks+block01+i])/Z/plmd.B;
    }
  }
}

#else

#define BLOCK 64
#define BATCH 16

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
// From http://stackoverflow.com/questions/16077464/atomicadd-for-real-on-gpu
// And https://stackoverflow.com/questions/37566987/cuda-atomicadd-for-doubles-definition-error
__device__ static inline
double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__device__ static inline
int rectify_modulus(int a,int b)
{
  int c=a%b;
  c-=(c>=b?b:0);
  c+=(c<0?b:0);
  return c;
}

template <typename real_type>
__device__ inline
void reduce(real_type input,real_type *shared,real_type *global)
{
  real_type local=input;
  local+=__shfl_down_sync(0xFFFFFFFF,local,1);
  local+=__shfl_down_sync(0xFFFFFFFF,local,2);
  local+=__shfl_down_sync(0xFFFFFFFF,local,4);
  local+=__shfl_down_sync(0xFFFFFFFF,local,8);
  local+=__shfl_down_sync(0xFFFFFFFF,local,16);
  __syncthreads();
  if ((0x1F & threadIdx.x)==0) {
    shared[threadIdx.x>>5]=local;
  }
  __syncthreads();
  local=0;
  if (threadIdx.x < (blockDim.x>>5)) {
    local=shared[threadIdx.x];
  }
  if (threadIdx.x < 32) {
    if (blockDim.x>=64) local+=__shfl_down_sync(0xFFFFFFFF,local,1);
    if (blockDim.x>=128) local+=__shfl_down_sync(0xFFFFFFFF,local,2);
    if (blockDim.x>=256) local+=__shfl_down_sync(0xFFFFFFFF,local,4);
    if (blockDim.x>=512) local+=__shfl_down_sync(0xFFFFFFFF,local,8);
    if (blockDim.x>=1024) local+=__shfl_down_sync(0xFFFFFFFF,local,16);
  }
  if (threadIdx.x==0) {
    atomicAdd(global,local);
  }
}

__device__ inline
int sort(int i,int *sortBuffer)
{
  int i1,i2;
  int direction,otherThreadIdx,iother,bswitch;
  int itmp=i;
  int Ztmp=threadIdx.x;

  // Bitonic sort
  for (i1=1; i1<BLOCK; i1*=2) {
    direction=(((2*i1)&threadIdx.x)!=0); // 0 ascending, 1 descending
    for (i2=i1; i2>0; i2/=2) {
      otherThreadIdx=(threadIdx.x^i2);
      if (i2<32) {
        iother=__shfl_xor_sync(0xFFFFFFFF,itmp,i2);
        bswitch=(((otherThreadIdx>threadIdx.x)==(iother>itmp))==direction);
        bswitch=(iother==itmp?0:bswitch);
        itmp=__shfl_sync(0xFFFFFFFF,itmp,threadIdx.x^(i2*bswitch));
        Ztmp=__shfl_sync(0xFFFFFFFF,Ztmp,threadIdx.x^(i2*bswitch));
      } else {
        sortBuffer[threadIdx.x]=itmp;
        __syncthreads();
        iother=sortBuffer[otherThreadIdx];
        bswitch=(((otherThreadIdx>threadIdx.x)==(iother>itmp))==direction);
        bswitch=(iother==itmp?0:bswitch);
        itmp=sortBuffer[threadIdx.x^(i2*bswitch)];
        __syncthreads();
        sortBuffer[threadIdx.x]=Ztmp;
        __syncthreads();
        Ztmp=sortBuffer[threadIdx.x^(i2*bswitch)];
        __syncthreads();
      }
    }
  }
  // threadIdx.x is destination, Ztmp is source
  sortBuffer[Ztmp]=threadIdx.x;
  __syncthreads();
  Ztmp=sortBuffer[threadIdx.x];
  // Now threadIdx.x is source, and Ztmp is destination
  __syncthreads();
  sortBuffer[threadIdx.x]=itmp;
  __syncthreads();

  return Ztmp;
}

__device__ inline
void reduce_sorted(reala input,int *sortBuffer,int iDest,reala *sreduceBuffer,reala *shared)
{
  int i1;

  int itmp=sortBuffer[threadIdx.x];
  sreduceBuffer[iDest]=input;
  __syncthreads();

  // Reduction
  for (i1=1; i1<BLOCK; i1*=2) {
    if ((threadIdx.x&i1) && (threadIdx.x&(i1-1))==0) {
      if (itmp==sortBuffer[threadIdx.x-i1]) {
        sreduceBuffer[threadIdx.x-i1]+=sreduceBuffer[threadIdx.x];
      } else {
        shared[itmp]+=sreduceBuffer[threadIdx.x];
      }
    }
    __syncthreads();
  }
  if (threadIdx.x==0) {
    shared[itmp]+=sreduceBuffer[threadIdx.x];
  }
  __syncthreads();
}

__global__
void regularize_function(struct_plmd plmd)
{
  int t=blockIdx.x*blockDim.x+threadIdx.x;
  int i=t;
  reala L;
  __shared__ reala reduceBuffer[BLOCK>>5];

  L=0.0;

  if (i<plmd.nblocks) {
    L+=0.5*plmd.kh*plmd.h_d[i]*plmd.h_d[i];
    // real h=tex1Dfetch<real>(plmd.h_tex,i);
    // L+=0.5*plmd.kh*h*h;
  }

  if (i<plmd.nblocks*plmd.nblocks) {
    L+=0.5*plmd.kJ*plmd.J_d[i]*plmd.J_d[i];
    // real J=tex1Dfetch<real>(plmd.J_tex,i);
    // L+=0.5*plmd.kJ*J*J;
  }

  reduce(L,reduceBuffer,plmd.L_d);
}

__global__
void regularize_function_line(real s,struct_plmd plmd)
{
  int t=blockIdx.x*blockDim.x+threadIdx.x;
  int i=t;
  reala L;
  reala dLds;
  __shared__ reala reduceBuffer[BLOCK>>5];

  L=0.0;
  dLds=0.0;

  if (i<plmd.nblocks) {
    L+=0.5*plmd.kh*(plmd.h_d[i]+s*plmd.dhds_d[i])*(plmd.h_d[i]+s*plmd.dhds_d[i]);
    dLds+=plmd.kh*(plmd.h_d[i]+s*plmd.dhds_d[i])*plmd.dhds_d[i];
    // real h=tex1Dfetch<real>(plmd.h_tex,i);
    // real dhds=tex1Dfetch<real>(plmd.dhds_tex,i);
    // L+=0.5*plmd.kh*(h+s*dhds)*(h+s*dhds);
    // dLds+=plmd.kh*(h+s*dhds)*dhds;
  }

  if (i<plmd.nblocks*plmd.nblocks) {
    L+=0.5*plmd.kJ*(plmd.J_d[i]+s*plmd.dJds_d[i])*(plmd.J_d[i]+s*plmd.dJds_d[i]);
    dLds+=plmd.kJ*(plmd.J_d[i]+s*plmd.dJds_d[i])*plmd.dJds_d[i];
    // real J=tex1Dfetch<real>(plmd.J_tex,i);
    // real dJds=tex1Dfetch<real>(plmd.dJds_tex,i);
    // L+=0.5*plmd.kJ*(J+s*dJds)*(J+s*dJds);
    // dLds+=plmd.kJ*(J+s*dJds)*dJds;
  }

  reduce(L,reduceBuffer,plmd.L_d);
  reduce(dLds,reduceBuffer,plmd.dLds_d);
}

__global__
void partition_function(struct_plmd plmd)
{
  // int t=blockIdx.x*blockDim.x+threadIdx.x;
  int site=blockIdx.y;
  int ib,b;
  int i,j;
  real heff;
  int block0,nsubs;

  block0=plmd.block0_d[site];
  nsubs=plmd.nsubs_d[site];
  for (ib=0; ib<BATCH; ib++) {
    b=blockIdx.x*blockDim.x*BATCH+blockDim.x*ib+threadIdx.x;
    if (b<plmd.BiB) {
      for (i=0; i<nsubs; i++) {
        heff=plmd.h_d[block0+i];
        // heff=tex1Dfetch<real>(plmd.h_tex,block0+i);
        for (j=0; j<plmd.nsites; j++) {
          heff+=plmd.J_d[plmd.nblocks*(block0+i) + plmd.Seq_d[b*plmd.nsites+j]];
          // heff+=tex1Dfetch<real>(plmd.J_tex,plmd.nblocks*(block0+i) + plmd.Seq_d[b*plmd.nsites+j]);
        }
        plmd.heff_d[b*plmd.nblocks+block0+i]=heff;
      }
    }
  }
}

__global__
void partition_function_line(struct_plmd plmd)
{
  // int t=blockIdx.x*blockDim.x+threadIdx.x;
  int site=blockIdx.y;
  int ib,b;
  int i,j;
  real heff;
  int block0,nsubs;

  block0=plmd.block0_d[site];
  nsubs=plmd.nsubs_d[site];
  for (ib=0; ib<BATCH; ib++) {
    b=blockIdx.x*blockDim.x*BATCH+blockDim.x*ib+threadIdx.x;
    if (b<plmd.BiB) {
      for (i=0; i<nsubs; i++) {
        heff=plmd.dhds_d[block0+i];
        // heff=tex1Dfetch<real>(plmd.dhds_tex,block0+i);
        for (j=0; j<plmd.nsites; j++) {
          heff+=plmd.dJds_d[plmd.nblocks*(block0+i) + plmd.Seq_d[b*plmd.nsites+j]];
          // heff+=tex1Dfetch<real>(plmd.dJds_tex,plmd.nblocks*(block0+i) + plmd.Seq_d[b*plmd.nsites+j]);
        }
        plmd.dheffds_d[b*plmd.nblocks+block0+i]=heff;
      }
    }
  }
}

__global__
void evaluate_function(struct_plmd plmd)
{
  int site=blockIdx.y;
  int ib,b;
  int i;
  real Z;
  int block0,nsubs;
  reala L;
  __shared__ reala reduceBuffer[BLOCK>>5];

  L=0.0;

  block0=plmd.block0_d[site];
  nsubs=plmd.nsubs_d[site];
  for (ib=0; ib<BATCH; ib++) {
    b=blockIdx.x*blockDim.x*BATCH+blockDim.x*ib+threadIdx.x;
    if (b<plmd.BiB) {
      L+=-plmd.heff_d[b*plmd.nblocks+plmd.Seq_d[b*plmd.nsites+site]];
      Z=0.0;
      for (i=0; i<nsubs; i++) {
        Z+=exp(plmd.heff_d[b*plmd.nblocks+block0+i]);
      }
      L+=log(Z);
    }
  }

  L/=plmd.B;
  reduce(L,reduceBuffer,plmd.L_d);
}

__global__
void evaluate_function_line(real s,struct_plmd plmd)
{
  int site=blockIdx.y;
  int ib,b;
  int i;
  real m1,Z,w;
  int block0,nsubs;
  reala dLds, L;
  __shared__ reala reduceBuffer[BLOCK>>5];

  L=0.0;
  dLds=0.0;

  block0=plmd.block0_d[site];
  nsubs=plmd.nsubs_d[site];
  for (ib=0; ib<BATCH; ib++) {
    b=blockIdx.x*blockDim.x*BATCH+blockDim.x*ib+threadIdx.x;
    if (b<plmd.BiB) {
      L+=-plmd.heff_d[b*plmd.nblocks+plmd.Seq_d[b*plmd.nsites+site]];
      L+=-s*plmd.dheffds_d[b*plmd.nblocks+plmd.Seq_d[b*plmd.nsites+site]];
      dLds+=-plmd.dheffds_d[b*plmd.nblocks+plmd.Seq_d[b*plmd.nsites+site]];
      m1=0.0;
      Z=0.0;
      for (i=0; i<nsubs; i++) {
        w=exp(plmd.heff_d[b*plmd.nblocks+block0+i]+s*plmd.dheffds_d[b*plmd.nblocks+block0+i]);
        Z+=w;
        m1+=plmd.dheffds_d[b*plmd.nblocks+block0+i]*w;
      }
      L+=log(Z);
      dLds+=m1/Z;
    }
  }

  L/=plmd.B;
  dLds/=plmd.B;
  reduce(L,reduceBuffer,plmd.L_d);
  reduce(dLds,reduceBuffer,plmd.dLds_d);
}

__global__
void regularize_gradient(struct_plmd plmd)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;

  if (i<plmd.nblocks) {
    plmd.dLdh_d[i]=plmd.kh*plmd.h_d[i];
    // real h=tex1Dfetch<real>(plmd.h_tex,i);
    // plmd.dLdh_d[i]=plmd.kh*h;
  }
  if (i<plmd.nblocks*plmd.nblocks) {
    plmd.dLdJ_d[i]=plmd.kJ*plmd.J_d[i];
    // real J=tex1Dfetch<real>(plmd.J_tex,i);
    // plmd.dLdJ_d[i]=plmd.kJ*J;
  }
}

__global__
void evaluate_gradient_h(struct_plmd plmd)
{
  int site=blockIdx.y;
  int ib,b;
  int i;
  int Seq;
  real Z;
  int block0,nsubs;
  __shared__ reala reduceBuffer[BLOCK>>5];
  extern __shared__ reala dLdh_s[];
  reala dLdh;

  block0=plmd.block0_d[site];
  nsubs=plmd.nsubs_d[site];
  if (threadIdx.x<nsubs) {
    dLdh_s[threadIdx.x]=0;
  }
  for (ib=0; ib<BATCH; ib++) {
    b=blockIdx.x*blockDim.x*BATCH+blockDim.x*ib+threadIdx.x;
    if (b<plmd.BiB) Seq=plmd.Seq_d[b*plmd.nsites+site]-block0;
    Z=0;
    for (i=0; i<nsubs; i++) {
      if (b<plmd.BiB) Z+=exp(plmd.heff_d[b*plmd.nblocks+block0+i]);
    }
    for (i=0; i<nsubs; i++) {
      dLdh=0;
      if (b<plmd.BiB) dLdh=exp(plmd.heff_d[b*plmd.nblocks+block0+i])/Z-(i==Seq);
      reduce(dLdh,reduceBuffer,&dLdh_s[i]);
    }
  }

  __syncthreads();
  if (threadIdx.x<nsubs) {
    dLdh_s[threadIdx.x]/=plmd.B;
    atomicAdd(&plmd.dLdh_d[block0+threadIdx.x],dLdh_s[threadIdx.x]);
  }
}

__global__
void evaluate_gradient_J(struct_plmd plmd)
{
  int site1=blockIdx.y;
  int site2=blockIdx.z;
  int ib,b;
  int i;
  int Seq1,Seq2;
  int iDest;
  real Z;
  int block01,block02,nsubs1,nsubs2;
  __shared__ reala sreduceBuffer[BLOCK];
  __shared__ int sortBuffer[BLOCK];
  extern __shared__ reala dLdJ_s[];
  reala dLdJ;

  if (site1==site2) return;

  block01=plmd.block0_d[site1];
  block02=plmd.block0_d[site2];
  nsubs1=plmd.nsubs_d[site1];
  nsubs2=plmd.nsubs_d[site2];
  if (threadIdx.x<nsubs1*nsubs2) {
    dLdJ_s[threadIdx.x]=0;
  }
  for (ib=0; ib<BATCH; ib++) {
    b=blockIdx.x*blockDim.x*BATCH+blockDim.x*ib+threadIdx.x;
    Seq1=0;
    Seq2=0;
    if (b<plmd.BiB) Seq1=plmd.Seq_d[b*plmd.nsites+site1]-block01;
    if (b<plmd.BiB) Seq2=plmd.Seq_d[b*plmd.nsites+site2]-block02;
    // iSource=threadIdx.x;
    iDest=sort(Seq2,sortBuffer);
    Z=0;
    for (i=0; i<nsubs1; i++) {
      if (b<plmd.BiB) Z+=exp(plmd.heff_d[b*plmd.nblocks+block01+i]);
    }
    for (i=0; i<nsubs1; i++) {
      dLdJ=0;
      if (b<plmd.BiB) dLdJ=exp(plmd.heff_d[b*plmd.nblocks+block01+i])/Z-(i==Seq1);
      reduce_sorted(dLdJ,sortBuffer,iDest,sreduceBuffer,&dLdJ_s[i*nsubs2]);
    }
  }

  __syncthreads();
  if (threadIdx.x<nsubs1*nsubs2) {
    dLdJ_s[threadIdx.x]/=plmd.B;
    Seq1=threadIdx.x/nsubs2;
    Seq2=rectify_modulus(threadIdx.x,nsubs2);
    atomicAdd(&plmd.dLdJ_d[plmd.nblocks*(block01+Seq1)+block02+Seq2],dLdJ_s[threadIdx.x]);
  }
}
#endif

void evaluateL(struct_plmd *plmd)
{
  copyout_hJ(plmd);
#ifdef PLMGPU
  hipMemcpy(plmd->h_d,plmd->h,plmd->nblocks*sizeof(real),hipMemcpyHostToDevice);
  hipMemcpy(plmd->J_d,plmd->J,plmd->nblocks*plmd->nblocks*sizeof(real),hipMemcpyHostToDevice);

  hipMemset(plmd->L_d,0,sizeof(reala));
  regularize_function<<<(plmd->nblocks*plmd->nblocks+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0]);
#else
  regularize_function(plmd[0]);
#endif

  if (plmd->B>plmd->BiBmax) {
    rewind(plmd->Seq_fp);
  }
  for (plmd->iB=0; plmd->iB<(plmd->B+plmd->BiBmax-1)/plmd->BiBmax; plmd->iB++) {
    plmd->BiB=plmd->B-plmd->iB*plmd->BiBmax;
    plmd->BiB=(plmd->BiB>plmd->BiBmax?plmd->BiBmax:plmd->BiB);
    if (plmd->B>plmd->BiBmax) {
      fread(plmd->Seq,sizeof(int),plmd->BiB*plmd->nsites,plmd->Seq_fp);
      hipMemcpy(plmd->Seq_d,plmd->Seq,plmd->BiB*plmd->nsites*sizeof(int),hipMemcpyHostToDevice);
    }
#ifdef PLMGPU
    {
      dim3 block_dim(BLOCK);
      dim3 grid_dim((plmd->BiB+BLOCK*BATCH+1)/(BLOCK*BATCH),plmd->nsites,1);
      partition_function<<<grid_dim,block_dim>>>(plmd[0]);
      evaluate_function<<<grid_dim,block_dim>>>(plmd[0]);
    }
#else
    #pragma omp parallel
    {
      int ID=omp_get_thread_num();
      int NID=omp_get_max_threads();
      for (int site=ID; site<plmd->nsites; site+=NID) {
        partition_function(site,plmd->block0[site],plmd[0]);
        evaluate_function(site,plmd->block0[site],plmd[0]);
      }
    }
#endif
  }

#ifdef PLMGPU
  hipMemcpy(plmd->L,plmd->L_d,sizeof(reala),hipMemcpyDeviceToHost);
#endif
}

void evaluateL_line(real s,struct_plmd *plmd)
{
#ifdef PLMGPU
  hipMemset(plmd->L_d,0,sizeof(reala));
  hipMemset(plmd->dLds_d,0,sizeof(reala));
  regularize_function_line<<<(plmd->nblocks*plmd->nblocks+BLOCK-1)/BLOCK,BLOCK>>>(s,plmd[0]);
#else
  regularize_function_line(s,plmd[0]);
#endif

  if (plmd->B>plmd->BiBmax) {
    rewind(plmd->Seq_fp);
  }
  for (plmd->iB=0; plmd->iB<(plmd->B+plmd->BiBmax-1)/plmd->BiBmax; plmd->iB++) {
    plmd->BiB=plmd->B-plmd->iB*plmd->BiBmax;
    plmd->BiB=(plmd->BiB>plmd->BiBmax?plmd->BiBmax:plmd->BiB);
    if (plmd->B>plmd->BiBmax) {
      fread(plmd->Seq,sizeof(int),plmd->BiB*plmd->nsites,plmd->Seq_fp);
#ifdef PLMGPU
      hipMemcpy(plmd->Seq_d,plmd->Seq,plmd->BiB*plmd->nsites*sizeof(int),hipMemcpyHostToDevice);
      dim3 block_dim(BLOCK);
      dim3 grid_dim((plmd->BiB+BLOCK*BATCH+1)/(BLOCK*BATCH),plmd->nsites,1);
      partition_function<<<grid_dim,block_dim>>>(plmd[0]);
      partition_function_line<<<grid_dim,block_dim>>>(plmd[0]);
#else
      #pragma omp parallel
      {
        int ID=omp_get_thread_num();
        int NID=omp_get_max_threads();
        for (int site=ID; site<plmd->nsites; site+=NID) {
          partition_function(site,plmd->block0[site],plmd[0]);
          partition_function_line(site,plmd->block0[site],plmd[0]);
        }
      }
#endif
    }
#ifdef PLMGPU
    {
      dim3 block_dim(BLOCK);
      dim3 grid_dim((plmd->BiB+BLOCK*BATCH+1)/(BLOCK*BATCH),plmd->nsites,1);
      evaluate_function_line<<<grid_dim,block_dim>>>(s,plmd[0]);
    }
#else
    #pragma omp parallel
    {
      int ID=omp_get_thread_num();
      int NID=omp_get_max_threads();
      for (int site=ID; site<plmd->nsites; site+=NID) {
        evaluate_function_line(s,site,plmd->block0[site],plmd[0]);
      }
    }
#endif
  }

#ifdef PLMGPU
  hipMemcpy(plmd->L,plmd->L_d,sizeof(reala),hipMemcpyDeviceToHost);
  hipMemcpy(plmd->dLds,plmd->dLds_d,sizeof(reala),hipMemcpyDeviceToHost);
#endif
}

void evaluatedLdx(struct_plmd *plmd)
{
#ifdef PLMGPU
  regularize_gradient<<<(plmd->nblocks*plmd->nblocks+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0]);
#else
  regularize_gradient(plmd[0]);
#endif

  if (plmd->B>plmd->BiBmax) {
    rewind(plmd->Seq_fp);
  }
  for (plmd->iB=0; plmd->iB<(plmd->B+plmd->BiBmax-1)/plmd->BiBmax; plmd->iB++) {
    plmd->BiB=plmd->B-plmd->iB*plmd->BiBmax;
    plmd->BiB=(plmd->BiB>plmd->BiBmax?plmd->BiBmax:plmd->BiB);
    if (plmd->B>plmd->BiBmax) {
      fread(plmd->Seq,sizeof(int),plmd->BiB*plmd->nsites,plmd->Seq_fp);
#ifdef PLMGPU
      hipMemcpy(plmd->Seq_d,plmd->Seq,plmd->BiB*plmd->nsites*sizeof(int),hipMemcpyHostToDevice);
      dim3 block_dim(BLOCK);
      dim3 grid_dim((plmd->BiB+BLOCK*BATCH+1)/(BLOCK*BATCH),plmd->nsites,1);
      partition_function<<<grid_dim,block_dim>>>(plmd[0]);
#else
      #pragma omp parallel
      {
        int ID=omp_get_thread_num();
        int NID=omp_get_max_threads();
        for (int site=ID; site<plmd->nsites; site+=NID) {
          partition_function(site,plmd->block0[site],plmd[0]);
        }
      }
#endif
    }
#ifdef PLMGPU
    {
      dim3 block_dim(BLOCK);
      dim3 grid_dim((plmd->BiB+BLOCK*BATCH+1)/(BLOCK*BATCH),plmd->nsites,1);
      evaluate_gradient_h<<<grid_dim,block_dim>>>(plmd[0]);
    }
    {
      dim3 block_dim(BLOCK);
      dim3 grid_dim((plmd->BiB+BLOCK*BATCH+1)/(BLOCK*BATCH),plmd->nsites,plmd->nsites);
      int shmem=plmd->nsubsmax*plmd->nsubsmax*sizeof(reala);
      evaluate_gradient_J<<<grid_dim,block_dim,shmem>>>(plmd[0]);
    }
#else
    #pragma omp parallel
    {
      int ID=omp_get_thread_num();
      int NID=omp_get_max_threads();
      for (int site1=ID; site1<plmd->nsites; site1+=NID) {
        int block01=plmd->block0[site1];
        evaluate_gradient_h(site1,block01,plmd[0]);
        for (int site2=0; site2<plmd->nsites; site2++) {
          int block02=plmd->block0[site2];
          if (site1 != site2) {
            evaluate_gradient_J(site1,site2,block01,block02,plmd[0]);
          }
        }
      }
    }
#endif
  }

#ifdef PLMGPU
  hipMemcpy(plmd->dLdh,plmd->dLdh_d,plmd->nblocks*sizeof(reala),hipMemcpyDeviceToHost);
  hipMemcpy(plmd->dLdJ,plmd->dLdJ_d,plmd->nblocks*plmd->nblocks*sizeof(reala),hipMemcpyDeviceToHost);
#endif

  copyback_gradient(plmd);
}

void resetHinv(struct_plmd *plmd)
{
  int i;
  // N^2 Hinv
  // for (i=0; i<plmd->Jend*plmd->Jend; i++) {
  //   plmd->Hinv[i]=0.0;
  // }
  // for (i=0; i<plmd->Jend; i++) {
  //   plmd->Hinv[(plmd->Jend+1)*i]=1.0;
  // }
  // N^1 Hinv
  for (i=0; i<plmd->Jend; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }
}

void updateHinv(struct_plmd *plmd)
{
  /* // Begin N^2 Hinv
  int i,j;
  real DxDg,DgHinvDg;
  real c1,c2;

  DxDg=0.0;
  for (i=0;i<plmd->Jend;i++) {
    // Put Delta x and Delta dLdx in x0 and dLdx0, which hold previous values
    plmd->x0[i]=plmd->x[i]-plmd->x0[i];
    plmd->dLdx0[i]=plmd->dLdx[i]-plmd->dLdx0[i];
    DxDg+=plmd->x0[i]*plmd->dLdx0[i];
  }

  DgHinvDg=0.0;
  for (i=0; i<plmd->Jend; i++) {
    plmd->hi[i]=0.0;
    for (j=0; j<plmd->Jend; j++) {
      // put Hinv * Delta dLdx in hi (the search direction) as a buffer
      plmd->hi[i]+=plmd->Hinv[i*plmd->Jend+j]*plmd->dLdx0[j];
    }
    DgHinvDg+=plmd->hi[i]*plmd->dLdx0[i];
  }
  c1=(1.0+DgHinvDg/DxDg)/DxDg;
  c2=-1.0/DxDg;

  for (i=0; i<plmd->Jend; i++) {
    for (j=i; j<plmd->Jend; j++) {
      plmd->Hinv[i*plmd->Jend+j]+=c1*(plmd->x0[i]*plmd->x0[j])+c2*(plmd->hi[i]*plmd->x0[j]+plmd->x0[i]*plmd->hi[j]);
      plmd->Hinv[j*plmd->Jend+i]=plmd->Hinv[i*plmd->Jend+j];
    }
  }
  
  //   dx=xf-xi;
  //   dd=df-di;
  //   Hinv=Hinv+(1+(dd'*Hinv*dd)/(dx'*dd))*(dx*dx')/(dx'*dd)-((Hinv*dd*dx')+(Hinv*dd*dx')')/(dx'*dd);
  */ // End N^2 Hinv
  // Begin N^1 Hinv
  int i,j;

  if (plmd->Nmem<plmd->Nmemax) {
    plmd->Nmem++;
  }
  for (i=plmd->Nmem-1; i>0; i--) {
    for (j=0; j<plmd->Jend; j++) {
      plmd->d_x[i*plmd->Jend+j]=plmd->d_x[(i-1)*plmd->Jend+j];
      plmd->d_dLdx[i*plmd->Jend+j]=plmd->d_dLdx[(i-1)*plmd->Jend+j];
    }
    plmd->rho[i]=plmd->rho[i-1];
  }

  plmd->rho[0]=0;
  for (i=0; i<plmd->Jend; i++) {
    plmd->d_x[i]=plmd->x[i]-plmd->x0[i];
    plmd->d_dLdx[i]=plmd->dLdx[i]-plmd->dLdx0[i];
    plmd->rho[0]+=plmd->d_x[i]*plmd->d_dLdx[i];
  }
  plmd->rho[0]=1.0/plmd->rho[0];

  for (i=0; i<plmd->Jend; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }
  // End N^1 Hinv
}

void projectHinv(struct_plmd *plmd)
{
/* // Begin N^2 Hinv
  int i,j;
  real dLds;
  //   hi=-Hinv*df;
  for (i=0; i<plmd->Jend; i++) {
    plmd->hi[i]=0;
    for (j=0; j<plmd->Jend; j++) {
      plmd->hi[i]+=-plmd->Hinv[i*plmd->Jend+j]*plmd->dLdx[j];
    }
  }

  dLds=0;
  for (i=0; i<plmd->Jend; i++) {
    dLds+=plmd->hi[i]*plmd->dLdx[i];
  }

  if (dLds>0) {
    fprintf(stderr,"Bad direction, reset Hinv\n");
    for (i=0; i<plmd->Jend*plmd->Jend; i++) {
      plmd->Hinv[i]=0;
    }
    for (i=0; i<plmd->Jend; i++) {
      plmd->Hinv[i*(plmd->Jend+1)]=1;
      plmd->hi[i]=plmd->dLdx[i];
    }
  }
*/ // End N^2 Hinv
// Begin N^1 Hinv
  int i,j;

  for (i=0; i<plmd->Jend; i++) {
    plmd->hi[i]=plmd->dLdx[i];
  }
  for (i=0; i<plmd->Nmem; i++) {
    plmd->alpha[i]=0;
    for (j=0; j<plmd->Jend; j++) {
      plmd->alpha[i]+=plmd->d_x[i*plmd->Jend+j]*plmd->hi[j];
    }
    plmd->alpha[i]*=plmd->rho[i];
    for (j=0; j<plmd->Jend; j++) {
      plmd->hi[j]+=-plmd->alpha[i]*plmd->d_dLdx[i*plmd->Jend+j];
    }
  }
  /*
  // According to wikipedia, this is to ensure the step length is always about unity
  // https://en.wikipedia.org/wiki/Limited-memory_BFGS
  if (plmd->Nmem>0) {
    numer=0.0L
    denom=0.0L;
    for (i=0; i<plmd->Jend; i++) {
      numer+=plmd->d_x[i]*plmd->hi[i];
      denom+=plmd->d_dLdx[i]*plmd->d_dLdx[i];
    }
    numer/=denom;
    for (i=0; i<plmd->Jend; i++) {
      plmd->hi[i]=numer*plmd->d_dLdx[i]; // This seems like a horrible idea, maybe wikipedia has a typo...
    }
  }
  */
  for (i=plmd->Nmem-1; i>=0; i--) {
    plmd->beta[i]=0;
    for (j=0; j<plmd->Jend; j++) {
      plmd->beta[i]+=plmd->d_dLdx[i*plmd->Jend+j]*plmd->hi[j];
    }
    plmd->beta[i]*=plmd->rho[i];
    for (j=0; j<plmd->Jend; j++) {
      plmd->hi[j]+=(plmd->alpha[i]-plmd->beta[i])*plmd->d_x[i*plmd->Jend+j];
    }
  }

  for (i=0; i<plmd->Jend; i++) {
    plmd->hi[i]*=-1;
  }
// End N^1 Hinv

  // New stuff
  copyout_gradient(plmd);
#ifdef PLMGPU
  hipMemcpy(plmd->dhds_d,plmd->dhds,plmd->nblocks*sizeof(real),hipMemcpyHostToDevice);
  hipMemcpy(plmd->dJds_d,plmd->dJds,plmd->nblocks*plmd->nblocks*sizeof(real),hipMemcpyHostToDevice);
#endif

  if (plmd->B>plmd->BiBmax) {
    rewind(plmd->Seq_fp);
  }
  for (plmd->iB=0; plmd->iB<(plmd->B+plmd->BiBmax-1)/plmd->BiBmax; plmd->iB++) {
    plmd->BiB=plmd->B-plmd->iB*plmd->BiBmax;
    plmd->BiB=(plmd->BiB>plmd->BiBmax?plmd->BiBmax:plmd->BiB);
    if (!(plmd->B>plmd->BiBmax)) {
#ifdef PLMGPU
      dim3 block_dim(BLOCK);
      dim3 grid_dim((plmd->BiB+BLOCK*BATCH+1)/(BLOCK*BATCH),plmd->nsites,1);
      partition_function_line<<<grid_dim,block_dim>>>(plmd[0]);
#else
      #pragma omp parallel
      {
        int ID=omp_get_thread_num();
        int NID=omp_get_max_threads();
        for (int site=ID; site<plmd->nsites; site+=NID) {
          partition_function_line(site,plmd->block0[site],plmd[0]);
        }
      }
#endif
    }
  }
}

void update_line(int step,struct_plmd *plmd)
{
  int i;
  reala a,b,c,s;
  real s1,s2,s3;
  reala L1,L2,L3;
  reala dLds1,dLds2,dLds3;
  reala L0;

  for (i=0; i<plmd->Jend; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }

  L0=plmd->L[0];

  s1=0.0;
  evaluateL_line(s1,plmd);
  L1=plmd->L[0];
  dLds1=plmd->dLds[0];
  if (dLds1>0) {
    fprintf(stderr,"Error, hi is pointing wrong way\n");
    exit(1);
  }
  
  s3=1.0;
  evaluateL_line(s3,plmd);
  L3=plmd->L[0];
  dLds3=plmd->dLds[0];

  while (dLds3<0 && s3<100000000L) {
    s2=s1-dLds1*(s3-s1)/(dLds3-dLds1);
    s3=(1.5*s2>8*s3 ? 8*s3 : 1.5*s2); // s2 is expected 0. Go past it by 50%, unless that's an increase of more than a factor of 8.
    evaluateL_line(s3,plmd);
    L3=plmd->L[0];
    dLds3=plmd->dLds[0];
  }

  s2=s1-dLds1*(s3-s1)/(dLds3-dLds1);
  evaluateL_line(s2,plmd);
  L2=plmd->L[0];
  dLds2=plmd->dLds[0];

  fprintf(stderr,"Step %4d s=%lg %lg %lg\n          L=%lg %lg %lg\n       dLds=%lg %lg %lg\n",
          step,(double) s1,(double) s2,(double) s3,
          (double) L1,(double) L2,(double) L3,
          (double) dLds1,(double) dLds2,(double) dLds3);
    
  for (i=0; i<15; i++) {
    if ((s2-s1)/s2<5e-7 || (s3-s2)/s2<5e-7 || dLds2==0) break;

    // Quadratic interpolation
    a=dLds1/((s1-s2)*(s1-s3));
    a+=dLds2/((s2-s1)*(s2-s3));
    a+=dLds3/((s3-s1)*(s3-s2));
    b=-dLds1*(s2+s3)/((s1-s2)*(s1-s3));
    b+=-dLds2*(s1+s3)/((s2-s1)*(s2-s3));
    b+=-dLds3*(s1+s2)/((s3-s1)*(s3-s2));
    c=dLds1*s2*s3/((s1-s2)*(s1-s3));
    c+=dLds2*s1*s3/((s2-s1)*(s2-s3));
    c+=dLds3*s1*s2/((s3-s1)*(s3-s2));
    s=(-b+sqrt(b*b-4*a*c))/(2*a);

    if (dLds2<0) {
      s1=s2;
      L1=L2;
      dLds1=dLds2;
    } else { // dLds2==0 already addressed above
      s3=s2;
      L3=L2;
      dLds3=dLds2;
    }

    if (s>s1 && s<s3) {
      // Use the earlier quadratic interpolation
      s2=s;
    } else {
      // Linear interpolation (secant method)
      fprintf(stdout,"Warning, fell back on linear interpolation\n");
      fprintf(stdout,"a=%lg b=%lg c=%lg s-=%lg s+=%lg s=%lg\n",(double)a,(double)b,(double)c,(double)((-b-sqrt(b*b-4*a*c))/(2*a)),(double)((-b+sqrt(b*b-4*a*c))/(2*a)),(double)s);
      s2=s1-dLds1*(s3-s1)/(dLds3-dLds1);
    }

    evaluateL_line(s2,plmd);
    L2=plmd->L[0];
    dLds2=plmd->dLds[0];

    fprintf(stderr,"Step %4d s=%lg %lg %lg\n          L=%lg %lg %lg\n       dLds=%lg %lg %lg\n",
            step,(double) s1,(double) s2,(double) s3,
            (double) L1,(double) L2,(double) L3,
            (double) dLds1,(double) dLds2,(double) dLds3);
  }

  fprintf(stderr,"Step %4d s=%lg %lg %lg\n          L=%lg %lg %lg\n       dLds=%lg %lg %lg\n",
          step,(double) s1,(double) s2,(double) s3,
          (double) L1,(double) L2,(double) L3,
          (double) dLds1,(double) dLds2,(double) dLds3);

  real stepLength2=0;

  for (i=0; i<plmd->Jend; i++) {
    plmd->x[i]=plmd->x0[i]+s2*plmd->hi[i];
    stepLength2+=(s2*plmd->hi[i])*(s2*plmd->hi[i]);
  }

  // fprintf(stderr,"Step %d smid1 %lg smid2 %lg L=%lg -> L1=%lg L2=%lg\n",step,(double) smid1,(double) smid2,(double) L0,(double) Lmid1,(double) Lmid2);
  fprintf(stderr,"Step %4d L=%24.16lf -> L2=%24.16lf, dL=%lg, step length=%lg\n",step,(double)L0,(double)L2,(double)(L2-L0),(double) sqrt(stepLength2));

  if (sqrt(stepLength2)<5e-7) plmd->done=true;
}

real lineL(real s,struct_plmd *plmd)
{
  int i;

  for (i=0; i<plmd->Jend; i++) {
    plmd->x[i]=plmd->x0[i]+s*plmd->hi[i];
  }

  evaluateL(plmd);
  return plmd->L[0];
}

void update(int step,struct_plmd *plmd)
{
  real smin,smid1,smid2,smax;
  real Lmin,Lmid1,Lmid2,Lmax;
  real L0;
  int i,ss;
  real phi=(1.0L+sqrtl(5.0L))/2.0L;

  for (i=0; i<plmd->Jend; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }
  smin=0.0L;
  smax=1.0L;

  // evaluateL(plmd);
  Lmin=plmd->L[0];
  L0=Lmin;

  Lmax=lineL(smax,plmd);
  // fprintf(stderr,"Step %d smax %g L=%g -> L=%g\n",step,smax,L0,Lmax);

  while (Lmax<Lmin && smax<100000000L) {
    smax*=2.0L;

    Lmax=lineL(smax,plmd);
    // fprintf(stderr,"Step %d smax %g L=%g -> L=%g\n",step,smax,L0,Lmax);
  }

  smid1=(smax-smin)/(phi*phi)+smin;
  Lmid1=lineL(smid1,plmd);

  smid2=-1.0L;
  
  for (ss=0; ss<25; ss++) {
    if (smid1<0.0L) {
      smid1=(smid2-smin)/phi+smin;
      Lmid1=lineL(smid1,plmd);
    } else {
      smid2=smax-(smax-smid1)/phi;
      Lmid2=lineL(smid2,plmd);
    }
    // fprintf(stderr,"Step %d smid1 %g smid2 %g L=%g -> L1=%g L2=%g\n",step,smid1,smid2,L0,Lmid1,Lmid2);
    fprintf(stderr,"Step %d smid1 %lg Lmin=%lg dL=%lg %lg %lg %lg\n",step,(double) smid1,(double) Lmin,(double) (Lmin-Lmin),(double) (Lmid1-Lmin),(double) (Lmid2-Lmin),(double) (Lmax-Lmin));

    if (Lmid1<=Lmid2) {
      smax=smid2;
      Lmax=Lmid2;
      smid2=smid1;
      Lmid2=Lmid1;
      smid1=-1.0L;
    } else {
      smin=smid1;
      Lmin=Lmid1;
      smid1=smid2;
      Lmid1=Lmid2;
      smid2=-1.0L;
    }
  }
  fprintf(stderr,"Step %d smid1 %lg smid2 %lg L=%lg -> L1=%lg L2=%lg\n",step,(double) smid1,(double) smid2,(double) L0,(double) Lmid1,(double) Lmid2);
}

/*
void lineL(double s,double *L,double *dLds,struct_plmd *plmd)
{
  int i;

  for (i=0; i<plmd->Jend; i++) {
    plmd->x[i]=plmd->x0[i]+s*plmd->hi[i];
  }

  evaluateL(plmd);
  evaluatedLdx(plmd);

  L[0]=plmd->L[0];

  dLds[0]=0;
  for (i=0; i<plmd->Jend; i++) {
    dLds[0]+=plmd->dLdx[i]*plmd->hi[i];
  }
}

void update(int step,struct_plmd *plmd)
{
  double smin,smid,smax;
  double Lmin,Lmid,Lmax;
  double dLdsmin,dLdsmid,dLdsmax;
  double L0;
  double d2Lds2;
  int i,ss;

  for (i=0; i<plmd->Jend; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }
  smin=0;
  smax=1;

  // evaluateL(plmd);
  Lmin=plmd->L[0];
  L0=Lmin;

  dLdsmin=0;
  for (i=0; i<plmd->Jend; i++) {
    dLdsmin+=plmd->dLdx[i]*plmd->hi[i];
  }

  lineL(smax,&Lmax,&dLdsmax,plmd);
  fprintf(stderr,"Step %d srange %g %g Lrange %g %g dLdsrange %g %g\n",step,smin,smax,Lmin,Lmax,dLdsmin,dLdsmax);

  for (ss=0; ss<10; ss++) {
    d2Lds2=(dLdsmax-dLdsmin)/(smax-smin);
    smid=smin-dLdsmin/d2Lds2;
    lineL(smid,&Lmid,&dLdsmid,plmd);

    if (dLdsmid>0) {
      smax=smid;
      Lmax=Lmid;
      dLdsmax=dLdsmid;
    } else if (dLdsmid<0) {
      smin=smid;
      Lmin=Lmid;
      dLdsmin=dLdsmid;
    } else {
      break;
    }
    fprintf(stderr,"Step %d srange %g %g Lrange %g %g dLdsrange %g %g\n",step,smin,smax,Lmin,Lmax,dLdsmin,dLdsmax);
  }
  fprintf(stderr,"Step %d smid %g L=%g -> L=%g\n",step,smid,L0,Lmid);
}
*/
/*
void initialize(struct_plmd *plmd)
{
  evaluateL(plmd);
  evaluatedLdx(plmd);

  projectHinv(plmd);
  update(-1,plmd);
}
*/

void itterate(int step,struct_plmd *plmd)
{
  evaluateL(plmd);
  evaluatedLdx(plmd);

  if (step==0) {
    resetHinv(plmd);
  } else {
    updateHinv(plmd);
  }

  projectHinv(plmd);
  update_line(step,plmd);
}

void run(struct_plmd *plmd)
{
  int s;

  //initialize(plmd);
  plmd->done=false;
  for (s=0; s<1000; s++) {
    itterate(s,plmd);
    if (plmd->done) break;
  }
}

void finish(struct_plmd *plmd,int argc, char *argv[])
{
  int i,j;
  FILE *fp;

  fp=fopen(argv[1],"w");
  for (i=0; i<plmd->nblocks; i++) {
    fprintf(fp," %lg",(double) plmd->h[i]);
  }
  fclose(fp);

  fp=fopen(argv[2],"w");
  for (i=0; i<plmd->nblocks; i++) {
    for (j=0; j<plmd->nblocks; j++) {
      fprintf(fp," %lg",(double) plmd->J[i*plmd->nblocks+j]);
    }
    fprintf(fp,"\n");
  }
  fclose(fp);

  free(plmd->nsubs);
  free(plmd->block2site);
  free(plmd->Seq);
  free(plmd->heff);

  free(plmd->x);
  free(plmd->dLdx);
  // free(plmd->Hinv);
  free(plmd->d_x);
  free(plmd->d_dLdx);
  free(plmd->rho);
  free(plmd->alpha);
  free(plmd->beta);
  free(plmd->hi);
  free(plmd->x0);
  free(plmd->dLdx0);

  // fclose(plmd->Seq_fp);

  /*hipDestroyTextureObject(plmd->h_tex);
  hipDestroyTextureObject(plmd->J_tex);
  hipDestroyTextureObject(plmd->dhds_tex);
  hipDestroyTextureObject(plmd->dJds_tex);*/
}

int main(int argc, char *argv[])
{
  struct_plmd *plmd;

  plmd = setup(argc,argv);
 
  run(plmd);

  finish(plmd,argc,argv);

  return 0;
}
