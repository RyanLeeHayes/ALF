#include "hip/hip_runtime.h"
// Written by Ryan Hayes 2017-06-20
// plmDCA algorithm from R470 - DOI: 10.1016/j.jcp.2014.07.024
// Quasi newton equations from https://www.rose-hulman.edu/~bryan/lottamath/quasinewton.pdf

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// #include <omp.h>

// #include <assert.h>

//   ID=omp_get_thread_num();
//   NID=omp_get_max_threads();

#include "lmalf.h"

#define MAXLENGTH 1024
#define BLOCK 512
#define kB 0.00198614L

#define NBINS 256
#define NBINS2 16

#define PROFILE true
#define MOMENT true

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
// From http://stackoverflow.com/questions/16077464/atomicadd-for-real-on-gpu
// And https://stackoverflow.com/questions/37566987/cuda-atomicadd-for-doubles-definition-error
__device__ static inline
double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

void printarray(real* x,int N,char* fnm)
{
  FILE *fp;
  real *xcpu;
  int i;

  fp=fopen(fnm,"w");
  xcpu=(real*)malloc(N*sizeof(real));
  hipMemcpy(xcpu,x,N*sizeof(real),hipMemcpyDeviceToHost);
  for (i=0; i<N; i++) {
    // fprintf(fp,"%g\n",xcpu[i]);
    fprintf(fp,"%22.15e\n",xcpu[i]);
  }
  free(xcpu);
  fclose(fp);
}

double randDouble()
{
  return (rand()+0.5)/(RAND_MAX+1.0);
}

void monte_carlo_Z(struct_plmd plmd)
{
  int ibeg,iend,Ns;
  int Neq=plmd.B/10;
  int Nmc=plmd.B;
  real *theta;
  int s,i,j;
  real b, st, norm;
  real thetaNew,eOld,eNew;

  theta=(real*) calloc(plmd.nblocks,sizeof(real));

  for (s=0; s<plmd.nsites; s++) {
    ibeg=plmd.block0[s];
    iend=plmd.block0[s+1];
    Ns=iend-ibeg;

    b=1;
    for (i=0; i<50; i++) {
      b=0.5*log(0.25*b*Ns*Ns*M_PI/2);
      if (!(b>0)) b=0;
    }

    theta[ibeg]=M_PI/2;
    for (i=ibeg+1; i<iend; i++) {
      theta[i]=3*M_PI/2;
    }

    for (i=-Neq; i<Nmc; i++) {
      if (i%Neq==0) {
        fprintf(stdout,"Partition Function Sample Step %d\n",i);
      }

      for (j=ibeg; j<iend; j++) {
        st=(-0.5*sin(theta[j])+0.5);
        eOld=-b*st*st*st*st;

        thetaNew=2*M_PI*randDouble();
        st=(-0.5*sin(thetaNew)+0.5);
        eNew=-b*st*st*st*st;

        if (exp(eOld-eNew)>randDouble()) {
          theta[j]=thetaNew;
        }
      }

      if (i>=0) {
        norm=0;
        for (j=ibeg; j<iend; j++) {
          norm+=exp(5.5*sin(theta[j]));
        }
        for (j=ibeg; j<iend; j++) {
          plmd.mc_lambda[plmd.nblocks*i+j]=exp(5.5*sin(theta[j]))/norm;
        }
      }
    }
  }

  free(theta);
}

struct_plmd* setup(int argc, char *argv[])
{
  struct_plmd *plmd;
  int si,sj,i,j,k,l;
  real k0;
  FILE *fp;
  char line[MAXLENGTH];

  if (argc<7) {
    fprintf(stderr,"Error: not enough input arguments\n");
    exit(1);
  }

  plmd=(struct_plmd*) malloc(sizeof(struct_plmd));

  fp=fopen("nsubs","r");
  plmd->nsites=0;
  while (fscanf(fp,"%d",&i)==1) {
    plmd->nsites++;
  }
  fclose(fp);

  fp=fopen("nsubs","r");
  i=0;
  plmd->nblocks=0;
  plmd->nsubs=(int*) calloc(plmd->nsites,sizeof(int));
  for(i=0; i<plmd->nsites; i++) {
    fscanf(fp,"%d",&(plmd->nsubs[i]));
    plmd->nblocks+=plmd->nsubs[i];
  }
  fclose(fp);

  plmd->block0=(int*) calloc(plmd->nsites+1,sizeof(int));
  plmd->block2site=(int*) calloc(plmd->nblocks,sizeof(int));
  k=0;
  for(i=0; i<plmd->nsites; i++) {
    plmd->block0[i]=k;
    for(j=0; j<plmd->nsubs[i]; j++) {
      plmd->block2site[k]=i;
      k++;
    }
  }
  plmd->block0[i]=k;

  hipMalloc(&(plmd->block0_d),(plmd->nsites+1)*sizeof(int));
  hipMemcpy(plmd->block0_d,plmd->block0,(plmd->nsites+1)*sizeof(int),hipMemcpyHostToDevice);

  i=sscanf(argv[2],"%d",&plmd->ms);
  if (i!=1) {
    fprintf(stderr,"Error, first argument must be a boolean flag for whether to use multisite coupling\n");
    exit(1);
  }

  i=sscanf(argv[3],"%d",&plmd->msprof);
  if (i!=1) {
    fprintf(stderr,"Error, second argument should indicate whether to use multisite profiles.\n");
    exit(1);
  }

  fp=fopen(argv[4],"r");
  for (plmd->B=0; fgets(line,MAXLENGTH,fp) != NULL; plmd->B++) {
    ;
  }
  fprintf(stdout,"%d frames\n",plmd->B); // DEBUG
  fclose(fp);

  plmd->lambda=(real*) calloc(plmd->B*plmd->nblocks,sizeof(real));
  plmd->ensweight=(real*) calloc(plmd->B,sizeof(real));
  plmd->mc_lambda=(real*) calloc(plmd->B*plmd->nblocks,sizeof(real));
  plmd->mc_ensweight=(real*) calloc(plmd->B,sizeof(real));

  fp=fopen(argv[4],"r");
  for (i=0;i<plmd->B;i++) {
    for (j=0;j<plmd->nblocks;j++) {
      double buffer;
      fscanf(fp,"%lf",&buffer);
      plmd->lambda[i*plmd->nblocks+j]=buffer;
    }
  }
  fclose(fp);

  monte_carlo_Z(plmd[0]);

  fp=fopen(argv[5],"r");
  for (i=0; i<plmd->B; i++) {
    double buffer;
    fscanf(fp,"%lf",&buffer);
    plmd->ensweight[i]=buffer;
    plmd->mc_ensweight[i]=1;
  }
  fclose(fp);

  if (argc>=8) {
    double criteria;
    i=sscanf(argv[7],"%lg",&criteria);
    fprintf(stdout,"Note, found seventh argument indicating halting criteria. Overwriting default value of 1.25e-3\n");
    if (i!=1) {
      fprintf(stderr,"Error, seventh argument should indicate halting criteria.\n");
      exit(1);
    }
    plmd->criteria=criteria;
  } else {
    plmd->critetia=1.25e-3;
  }

  hipMalloc(&(plmd->lambda_d),plmd->B*plmd->nblocks*sizeof(real));
  hipMalloc(&(plmd->mc_lambda_d),plmd->B*plmd->nblocks*sizeof(real));
  hipMalloc(&(plmd->ensweight_d),plmd->B*sizeof(real));
  hipMalloc(&(plmd->mc_ensweight_d),plmd->B*sizeof(real));

  hipMemcpy(plmd->lambda_d,plmd->lambda,plmd->B*plmd->nblocks*sizeof(real),hipMemcpyHostToDevice);
  hipMemcpy(plmd->mc_lambda_d,plmd->mc_lambda,plmd->B*plmd->nblocks*sizeof(real),hipMemcpyHostToDevice);
  hipMemcpy(plmd->ensweight_d,plmd->ensweight,plmd->B*sizeof(real),hipMemcpyHostToDevice);
  hipMemcpy(plmd->mc_ensweight_d,plmd->mc_ensweight,plmd->B*sizeof(real),hipMemcpyHostToDevice);



  // count nbias
  plmd->nbias=0;
  for (i=0; i<plmd->nsites; i++) {
    for (j=i; j<plmd->nsites; j++) {
      if (i==j) {
        plmd->nbias+=plmd->nsubs[i]+(5*plmd->nsubs[i]*(plmd->nsubs[i]-1))/2;
      } else if (plmd->ms==1) {
        plmd->nbias+=5*plmd->nsubs[i]*plmd->nsubs[j];
      } else if (plmd->ms==2) {
        plmd->nbias+=plmd->nsubs[i]*plmd->nsubs[j];
      }
    }
  }

  // count nprof
  plmd->nprof=0;
  for (i=0; i<plmd->nsites; i++) {
    for (j=i; j<plmd->nsites; j++) {
      if (i==j) {
        if (plmd->nsubs[i]==2) {
          plmd->nprof+=plmd->nsubs[i]+plmd->nsubs[i]*(plmd->nsubs[i]-1)/2;
        } else {
          plmd->nprof+=plmd->nsubs[i]+2*plmd->nsubs[i]*(plmd->nsubs[i]-1)/2;
        }
      } else if (plmd->msprof) {
        plmd->nprof+=plmd->nsubs[i]*plmd->nsubs[j];
      }
    }
  }

  // plmd->nx=plmd->nbias+plmd->nprof;
  plmd->nx=plmd->nbias;

  double temperature;
  sscanf(argv[1],"%lg",&temperature);
  plmd->kT=kB*temperature;

  // Set regularization constants
  real kp=1.0/(plmd->kT*plmd->kT);
  plmd->kx=(real*) calloc(plmd->nx,sizeof(real));
  plmd->xr=(real*) calloc(plmd->nx,sizeof(real));
  real *xr_x, *xr_s;
  // load starting values if needed for ms==1
  if (plmd->ms==1) {
    xr_x=(real*) calloc(plmd->nblocks*plmd->nblocks,sizeof(real));
    fp=fopen("x_prev.dat","r");
    for(i=0; i<plmd->nblocks; i++) {
      for(j=0; j<plmd->nblocks; j++) {
        double buffer;
        fscanf(fp,"%lf",&buffer);
        xr_x[i*plmd->nblocks+j]=buffer;
      }
    }
    fclose(fp);
    xr_s=(real*) calloc(plmd->nblocks*plmd->nblocks,sizeof(real));
    fp=fopen("s_prev.dat","r");
    for(i=0; i<plmd->nblocks; i++) {
      for(j=0; j<plmd->nblocks; j++) {
        double buffer;
        fscanf(fp,"%lf",&buffer);
        xr_s[i*plmd->nblocks+j]=buffer;
      }
    }
    fclose(fp);
  }
  // k0=1e-2; // 1.0/400;
  k0=kp/400;
  // k0=1;
  k=0;
  for (si=0; si<plmd->nsites; si++) {
    for (sj=si; sj<plmd->nsites; sj++) {
      if (si==sj) {
        for (i=0; i<plmd->nsubs[si]; i++) {
          plmd->kx[k++]=k0/4; // b
          for (j=i+1; j<plmd->nsubs[sj]; j++) {
            plmd->kx[k++]=k0/64; // c
            plmd->kx[k++]=k0/4; // x
            plmd->kx[k++]=k0/4; // x
            plmd->kx[k++]=k0/1; // s
            plmd->kx[k++]=k0/1; // s
          }
        }
      } else if (plmd->ms) {
        for (i=0; i<plmd->nsubs[si]; i++) {
          for (j=0; j<plmd->nsubs[sj]; j++) {
            plmd->kx[k++]=k0/4; // c
            if (plmd->ms==1) {
              plmd->xr[k]=xr_x[(plmd->block0[si]+i)*plmd->nblocks+plmd->block0[sj]+j]; // x
              plmd->kx[k++]=k0/0.25; // x
              plmd->xr[k]=xr_x[(plmd->block0[sj]+j)*plmd->nblocks+plmd->block0[si]+i]; // x
              plmd->kx[k++]=k0/0.25; // x
              plmd->xr[k]=xr_s[(plmd->block0[si]+i)*plmd->nblocks+plmd->block0[sj]+j]; // s
              plmd->kx[k++]=k0/0.25; // s
              plmd->xr[k]=xr_s[(plmd->block0[sj]+j)*plmd->nblocks+plmd->block0[si]+i]; // s
              plmd->kx[k++]=k0/0.25; // s
            }
          }
        }
      }
    }
  }
  if (plmd->ms==1) {
    free(xr_x);
    free(xr_s);
  }
  // No restraints on average profile values - treated implicitly now
  /*for (i=0; i<plmd->nprof; i++) {
    plmd->kx[k++]=0;
  }*/
  hipMalloc(&plmd->kx_d,plmd->nx*sizeof(real));
  hipMemcpy(plmd->kx_d,plmd->kx,plmd->nx*sizeof(real),hipMemcpyHostToDevice);
  hipMalloc(&plmd->xr_d,plmd->nx*sizeof(real));
  hipMemcpy(plmd->xr_d,plmd->xr,plmd->nx*sizeof(real),hipMemcpyHostToDevice);

  // plmd->kprofile=1.0/NBINS;
  real kp0=kp/NBINS;
  plmd->kprofile=(real*)calloc(NBINS*plmd->nprof,sizeof(real));
  k=0;
  for (si=0; si<plmd->nsites; si++) {
    for (sj=si; sj<plmd->nsites; sj++) {
      if (si==sj) { // Same site
        for (i=plmd->block0[si]; i<plmd->block0[si+1]; i++) {
          for (l=0; l<NBINS; l++) {
            plmd->kprofile[NBINS*k+l]=kp0;
            if (l==NBINS-1) plmd->kprofile[NBINS*k+l]*=(NBINS/4.0);
          }
          k++;
        }
        for (i=plmd->block0[si]; i<plmd->block0[si+1]; i++) {
          for (j=i+1; j<plmd->block0[sj+1]; j++) {
            for (l=0; l<NBINS; l++) {
              plmd->kprofile[NBINS*k+l]=kp0/((plmd->nsubs[si]-1)/2.0);
            }
            k++;
          }
        }
        if (plmd->nsubs[si]>2) {
          for (i=plmd->block0[si]; i<plmd->block0[si+1]; i++) {
            for (j=i+1; j<plmd->block0[sj+1]; j++) {
              for (l=0; l<NBINS; l++) {
                plmd->kprofile[NBINS*k+l]=kp0/((plmd->nsubs[si]-1)/2.0);
              }
              k++;
            }
          }
        }
      } else if (plmd->msprof) {
        for (i=plmd->block0[si]; i<plmd->block0[si+1]; i++) {
          for (j=plmd->block0[sj]; j<plmd->block0[sj+1]; j++) {
            for (l=0; l<NBINS; l++) {
              plmd->kprofile[NBINS*k+l]=kp0/(plmd->nsubs[si]*plmd->nsubs[sj]);
              if (l==NBINS-1) plmd->kprofile[NBINS*k+l]*=(NBINS/4.0);
            }
            k++;
          }
        }
      }
    }
  }
  hipMalloc(&plmd->kprofile_d,NBINS*plmd->nprof*sizeof(real));
  hipMemcpy(plmd->kprofile_d,plmd->kprofile,NBINS*plmd->nprof*sizeof(real),hipMemcpyHostToDevice);

  plmd->L=(real*) calloc(1,sizeof(real));
  plmd->dLds=(real*) calloc(1,sizeof(real));

  hipMalloc(&plmd->L_d,sizeof(real));
  hipMalloc(&plmd->dLds_d,sizeof(real));

  plmd->x=(real*) calloc(plmd->nx,sizeof(real));
  plmd->dLdx=(real*) calloc(plmd->nx,sizeof(real));
  hipMalloc(&plmd->dLdx_d,plmd->nx*sizeof(real));
  // plmd->Hinv=(real*) calloc(plmd->nx*plmd->nx,sizeof(real));
  plmd->Nmemax=50;
  plmd->Nmem=0;
  plmd->d_x=(real*) calloc(plmd->nx*plmd->Nmemax,sizeof(real));
  plmd->d_dLdx=(real*) calloc(plmd->nx*plmd->Nmemax,sizeof(real));
  plmd->rho=(real*) calloc(plmd->Nmemax,sizeof(real));
  plmd->alpha=(real*) calloc(plmd->nx*plmd->Nmemax,sizeof(real));
  plmd->beta=(real*) calloc(plmd->nx*plmd->Nmemax,sizeof(real));

  plmd->hi=(real*) calloc(plmd->nx,sizeof(real));
  plmd->x0=(real*) calloc(plmd->nx,sizeof(real));
  plmd->dLdx0=(real*) calloc(plmd->nx,sizeof(real));



  hipMalloc(&(plmd->E_d),plmd->B*sizeof(real));
  hipMalloc(&(plmd->dEds_d),plmd->B*sizeof(real));
  hipMalloc(&(plmd->mc_E_d),plmd->B*sizeof(real));
  hipMalloc(&(plmd->mc_dEds_d),plmd->B*sizeof(real));

  hipMalloc(&(plmd->weight_d),plmd->B*sizeof(real));
  hipMalloc(&(plmd->mc_weight_d),plmd->B*sizeof(real));

  hipMalloc(&(plmd->x_d),plmd->nx*sizeof(real));
  hipMalloc(&(plmd->dxds_d),plmd->nx*sizeof(real));

  hipMalloc(&(plmd->Z_d),plmd->nprof*NBINS*sizeof(real));
  hipMalloc(&(plmd->mc_Z_d),plmd->nprof*NBINS*sizeof(real));
  hipMalloc(&(plmd->Zprofile_d),plmd->nprof*NBINS*sizeof(real));
  hipMalloc(&(plmd->mc_Zprofile_d),plmd->nprof*NBINS*sizeof(real));

  hipMalloc(&(plmd->dLdZprofile_d),plmd->nprof*NBINS*sizeof(real));
  hipMalloc(&(plmd->mc_dLdZprofile_d),plmd->nprof*NBINS*sizeof(real));
  hipMalloc(&(plmd->dLdE_d),plmd->B*sizeof(real));
  hipMalloc(&(plmd->Gimp_d),plmd->nprof*NBINS*sizeof(real));
  hipMalloc(&(plmd->G_d),plmd->nprof*NBINS*sizeof(real));

  hipMalloc(&(plmd->Esum_d),sizeof(real));
  hipMalloc(&(plmd->dEdssum_d),sizeof(real));
  hipMalloc(&(plmd->mc_dEdssum_d),sizeof(real));
  hipMalloc(&(plmd->moments_d),plmd->nbias*sizeof(real));
  hipMalloc(&(plmd->mc_moments_d),plmd->nbias*sizeof(real));
  hipMalloc(&(plmd->sumensweight_d),sizeof(real));

  plmd->fplog=fopen("log.log","w");

  return plmd;
}

__device__
void reduce(real local,real* shared,real* global)
{
  int k;

  shared[threadIdx.x]=local;

  __syncthreads();

  for (k=1; k<BLOCK; k*=2) {
    if ((threadIdx.x % (2*k)) == 0) {
      shared[threadIdx.x]+=shared[threadIdx.x+k];
    }
    __syncthreads();
  }

  if (threadIdx.x==0) {
    atomicAdd(global,shared[0]);
  }
}

__device__
void reduceNBINS(real local,real* shared,real* global)
{
  int k;

  shared[threadIdx.x]=local;

  __syncthreads();

  for (k=1; k<NBINS; k*=2) {
    if ((threadIdx.x % (2*k)) == 0) {
      shared[threadIdx.x]+=shared[threadIdx.x+k];
    }
    __syncthreads();
  }

  if (threadIdx.x==0) {
    atomicAdd(global,shared[0]);
  }
}

__device__
void reduceBroadcast(real local,real* shared)
{
  int k;
  real buf;

  shared[threadIdx.x]=local;

  __syncthreads();

  for (k=1; k<NBINS; k*=2) {
    buf=0;
    if ((threadIdx.x^k)<NBINS) buf=shared[threadIdx.x^k];
    __syncthreads();
    shared[threadIdx.x]+=buf;
    __syncthreads();
  }
}

__device__
void reduceBitonicSort(int itmp,real Ztmp,int* iloc,real* Zloc,real* Zloc2,real* Zglobal)
{
  int i1,i2;
  int direction,otherThreadIdx,iother,bswitch;

  if (threadIdx.x<NBINS) {
    Zloc2[threadIdx.x]=0;
  }
  // Bitonic sort
  for (i1=1; i1<BLOCK; i1*=2) {
    direction=(((2*i1)&threadIdx.x)!=0); // 0 ascending, 1 descending
    for (i2=i1; i2>0; i2/=2) {
      otherThreadIdx=(threadIdx.x^i2);
      if (i2<32) {
        iother=__shfl_xor_sync(-1,itmp,i2);
        bswitch=(((otherThreadIdx>threadIdx.x)==(iother>itmp))==direction);
        bswitch=(iother==itmp?0:bswitch);
        itmp=__shfl_sync(-1,itmp,threadIdx.x^(i2*bswitch));
        Ztmp=__shfl_sync(-1,Ztmp,threadIdx.x^(i2*bswitch));
      } else {
        iloc[threadIdx.x]=itmp;
        Zloc[threadIdx.x]=Ztmp;
        __syncthreads();
        iother=iloc[otherThreadIdx];
        bswitch=(((otherThreadIdx>threadIdx.x)==(iother>itmp))==direction);
        bswitch=(iother==itmp?0:bswitch);
        itmp=iloc[threadIdx.x^(i2*bswitch)];
        Ztmp=Zloc[threadIdx.x^(i2*bswitch)];
        __syncthreads();
      }
    }
  }
  iloc[threadIdx.x]=itmp;
  Zloc[threadIdx.x]=Ztmp;
  __syncthreads();
  // Reduction
  for (i1=1; i1<BLOCK; i1*=2) {
    if ((threadIdx.x&i1) && (threadIdx.x&(i1-1))==0) {
      if (itmp==iloc[threadIdx.x-i1]) {
        Zloc[threadIdx.x-i1]+=Zloc[threadIdx.x];
      } else {
        Zloc2[itmp]+=Zloc[threadIdx.x];
      }
    }
    __syncthreads();
  }
  if (threadIdx.x==0) {
    Zloc2[itmp]+=Zloc[threadIdx.x];
  }
  __syncthreads();
  if (threadIdx.x<NBINS) {
    atomicAdd(&Zglobal[threadIdx.x],Zloc2[threadIdx.x]);
  }
}
/*
{
  if (itmp<NBINS) {
    atomicAdd(&Zglobal[itmp],Ztmp);
  }
}*/

__global__
void energykernel(struct_plmd plmd,real* x,real* lambda,real* energy)
{
  int b=blockIdx.x*blockDim.x+threadIdx.x;
  int s1,s2;
  int i1,i2;
  int k;
  real q1,q2;
  real E;

  lambda+=plmd.nblocks*b;

  if (b<plmd.B) {
    k=0;
    E=0;
    for (s1=0; s1<plmd.nsites; s1++) {
      for (s2=s1; s2<plmd.nsites; s2++) {
        if (s1==s2) { // Same site
          for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
            q1=lambda[i1];
            E+=x[k]*q1;
            k++;
            for (i2=i1+1; i2<plmd.block0_d[s1+1]; i2++) {
              q2=lambda[i2];
              E+=x[k]*q1*q2;
              k++;
              E+=x[k]*q2*(1-exp(-q1/0.18));
              k++;
              E+=x[k]*q1*(1-exp(-q2/0.18));
              k++;
              E+=x[k]*q2*(1-1/(q1/0.017+1));
              k++;
              E+=x[k]*q1*(1-1/(q2/0.017+1));
              k++;
            }
          }
        } else if (plmd.ms) { // Different sites
          for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
            q1=lambda[i1];
            for (i2=plmd.block0_d[s2]; i2<plmd.block0_d[s2+1]; i2++) {
              q2=lambda[i2];
              E+=x[k]*q1*q2;
              k++;
              if (plmd.ms==1) { // include extra terms
                E+=x[k]*q2*(1-exp(-q1/0.18));
                k++;
                E+=x[k]*q1*(1-exp(-q2/0.18));
                k++;
                E+=x[k]*q2*(1-1/(q1/0.017+1));
                k++;
                E+=x[k]*q1*(1-1/(q2/0.017+1));
                k++;
              }
            }
          }
        }
      }
    }
    energy[b]=E;
  }
}

__global__
void dotenergykernel(struct_plmd plmd,real sign,real* x,real* y,real* z)
{
  int b=blockIdx.x*blockDim.x+threadIdx.x;
  real xtmp;
  __shared__ real xloc[BLOCK];

  if (b<plmd.B) {
    xtmp=sign*x[b]*y[b];
  } else {
    xtmp=0;
  }
  reduce(xtmp,xloc,z);
}

__global__
void weightedenergykernel(struct_plmd plmd,real sign,real* lambda,real* weight,real* dEdx)
{
  int b=blockIdx.x*blockDim.x+threadIdx.x;
  int s1,s2;
  int i1,i2;
  int k;
  real q1,q2;
  real w,E;
  __shared__ real Eloc[BLOCK];

  lambda+=plmd.nblocks*b;

  w=0;
  q1=0;
  q2=0;

  if (b<plmd.B) {
    w=sign*weight[b];
  }

  k=0;
  for (s1=0; s1<plmd.nsites; s1++) {
    for (s2=s1; s2<plmd.nsites; s2++) {
      if (s1==s2) { // Same site
        for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
          if (b<plmd.B) q1=lambda[i1];
          E=w*q1;
          reduce(E,Eloc,&dEdx[k]);
          k++;
          for (i2=i1+1; i2<plmd.block0_d[s1+1]; i2++) {
            if (b<plmd.B) q2=lambda[i2];
            E=w*q1*q2;
            reduce(E,Eloc,&dEdx[k]);
            k++;
            E=w*q2*(1-exp(-q1/0.18));
            reduce(E,Eloc,&dEdx[k]);
            k++;
            E=w*q1*(1-exp(-q2/0.18));
            reduce(E,Eloc,&dEdx[k]);
            k++;
            E=w*q2*(1-1/(q1/0.017+1));
            reduce(E,Eloc,&dEdx[k]);
            k++;
            E=w*q1*(1-1/(q2/0.017+1));
            reduce(E,Eloc,&dEdx[k]);
            k++;
          }
        }
      } else if (plmd.ms) { // Different sites
        for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
          q1=lambda[i1];
          for (i2=plmd.block0_d[s2]; i2<plmd.block0_d[s2+1]; i2++) {
            q2=lambda[i2];
            E=w*q1*q2;
            reduce(E,Eloc,&dEdx[k]);
            k++;
            if (plmd.ms==1) { // include extra terms
              E=w*q2*(1-exp(-q1/0.18));
              reduce(E,Eloc,&dEdx[k]);
              k++;
              E=w*q1*(1-exp(-q2/0.18));
              reduce(E,Eloc,&dEdx[k]);
              k++;
              E=w*q2*(1-1/(q1/0.017+1));
              reduce(E,Eloc,&dEdx[k]);
              k++;
              E=w*q1*(1-1/(q2/0.017+1));
              reduce(E,Eloc,&dEdx[k]);
              k++;
            }
          }
        }
      }
    }
  }
}

__global__
void boltzmannkernel(struct_plmd plmd,real sign,real* energy,real s,real* denergyds,real* inweight,real* outweight,real* Z)
{
  int b=blockIdx.x*blockDim.x+threadIdx.x;
  real E;
  real w;
  __shared__ real Zloc[BLOCK];

  if (b<plmd.B) {
    w=inweight[b];
    E=energy[b];
    if (s) { // add the displacement if it is non-zero
      E+=s*denergyds[b];
    }
    w*=exp(-sign*E/plmd.kT);
    outweight[b]=w;
  } else {
    w=0;
  }

  if (Z) { // calculate the partition function if requested
    __syncthreads();
    reduce(w,Zloc,Z);
  }
}

__global__
void profilekernel(struct_plmd plmd,real* lambda,real* inweight,real* weightprofile,real* outweight,real* Zprofile)
{
  int b=blockIdx.x*blockDim.x+threadIdx.x;
  int s1,s2;
  int i1,i2;
  int k;
  real q1,q2;
  real w, wout;
  int itmp;
  real Ztmp;
  __shared__ int iloc[BLOCK];
  __shared__ real Zloc[BLOCK];
  __shared__ real Zloc2[NBINS+1];

  lambda+=plmd.nblocks*b;

  wout=0;

  if (b<plmd.B) {
    w=inweight[b];
  } else {
    w=0;
  }

  k=0;
  for (s1=0; s1<plmd.nsites; s1++) {
    for (s2=s1; s2<plmd.nsites; s2++) {
      if (s1==s2) { // Same site
        for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
          __syncthreads();
          itmp=NBINS;
          Ztmp=w;
          if (b<plmd.B) {
            q1=lambda[i1];
            itmp=(int)floor(q1*NBINS);
            // if (weightprofile) assert(w<=plmd.Zprofile_d[k*NBINS+itmp]);
            if (weightprofile) Ztmp*=weightprofile[k*NBINS+itmp];
          }
          if (outweight) wout+=Ztmp;
          if (Zprofile) reduceBitonicSort(itmp,Ztmp,iloc,Zloc,Zloc2,&Zprofile[k*NBINS]);
          k++;
        }

        for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
          for (i2=i1+1; i2<plmd.block0_d[s2+1]; i2++) {
            __syncthreads();
            itmp=NBINS;
            Ztmp=w;
            if (b<plmd.B) {
              q1=lambda[i1];
              q2=lambda[i2];
              if (q1+q2>0.8) {
                itmp=(int)floor(q1/(q1+q2)*NBINS);
                // if (weightprofile) assert(w<=plmd.Zprofile_d[k*NBINS+itmp]);
                if (weightprofile) Ztmp*=weightprofile[k*NBINS+itmp];
              } else { // WORKING - testing the next line
                Ztmp*=0;
              }
            }
            if (outweight) wout+=Ztmp;
            if (Zprofile) reduceBitonicSort(itmp,Ztmp,iloc,Zloc,Zloc2,&Zprofile[k*NBINS]);
            k++;
          }
        }

        if (plmd.block0_d[s1+1]-plmd.block0_d[s1]>2) {
          for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
            for (i2=i1+1; i2<plmd.block0_d[s2+1]; i2++) {
              __syncthreads();
              itmp=NBINS;
              Ztmp=w;
              if (b<plmd.B) {
                q1=lambda[i1];
                q2=lambda[i2];
                itmp=NBINS2*((int)floor(q1*NBINS2))+(int)floor(q2*NBINS2);
                // if (weightprofile) assert(w<=plmd.Zprofile_d[k*NBINS+itmp]);
                if (weightprofile) Ztmp*=weightprofile[k*NBINS+itmp];
              }
              if (outweight) wout+=Ztmp;
              if (Zprofile) reduceBitonicSort(itmp,Ztmp,iloc,Zloc,Zloc2,&Zprofile[k*NBINS]);
              k++;
            }
          }
        }
      } else if (plmd.msprof) {
        for (i1=plmd.block0_d[s1]; i1<plmd.block0_d[s1+1]; i1++) {
          for (i2=plmd.block0_d[s2]; i2<plmd.block0_d[s2+1]; i2++) {
            __syncthreads();
            itmp=NBINS;
            Ztmp=w;
            if (b<plmd.B) {
              q1=lambda[i1];
              q2=lambda[i2];
              itmp=NBINS2*((int)floor(q1*NBINS2))+(int)floor(q2*NBINS2);
              // if (weightprofile) assert(w<=plmd.Zprofile_d[k*NBINS+itmp]);
              if (weightprofile) Ztmp*=weightprofile[k*NBINS+itmp];
            }
            if (outweight) wout+=Ztmp;
            if (Zprofile) reduceBitonicSort(itmp,Ztmp,iloc,Zloc,Zloc2,&Zprofile[k*NBINS]);
            k++;
          }
        }
      }
    }
  }

  if (outweight) {
    if (b<plmd.B) {
      outweight[b]=wout;
    }
  }
}

__global__
void freeenergykernel(struct_plmd plmd,real* Zprofile,real* G,real* L,real* dLdZprof)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  // int iprof=blockIdx.x;
  real Ztmp, Gtmp, dGtmp, kprof, Gavg, Ltmp;
  __shared__ real Lloc[NBINS];

  if (i<plmd.nprof*NBINS) {
    Ztmp=Zprofile[i];
    Gtmp=-plmd.kT*log(Ztmp);
  } else {
    Ztmp=0;
    Gtmp=0;
  }

  if (G) {
    if (i<plmd.nprof*NBINS) {
      G[i]=Gtmp;
    }
  }
  if (L || dLdZprof) {
    if (Ztmp) {
      kprof=plmd.kprofile_d[i];
      dGtmp=Gtmp-plmd.Gimp_d[i];
    } else {
      kprof=0;
      dGtmp=0;
    }
    reduceBroadcast(kprof*dGtmp,Lloc);
    Gavg=Lloc[threadIdx.x];
    reduceBroadcast(kprof,Lloc);
    if (Lloc[threadIdx.x]) Gavg/=Lloc[threadIdx.x];
    if (Ztmp) dGtmp-=Gavg; // Contribution of Gavg to derivatives of L with respect to Gtmp magically cancels out
  }
  if (L) {
    Ltmp=0.5*kprof*dGtmp*dGtmp;
    reduceNBINS(Ltmp,Lloc,L);
  }
  if (dLdZprof) {
    if (Ztmp) {
      dLdZprof[i]=kprof*dGtmp/Ztmp;
    }
  }
}

__global__
void regularizeLkernel(struct_plmd plmd)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  real deltax, L;
  __shared__ real Lloc[BLOCK];

  if (i<plmd.nx) {
    deltax=plmd.x_d[i]-plmd.xr_d[i];
    L=0.5*plmd.kx_d[i]*deltax*deltax;
  } else {
    L=0;
  }

  reduce(L,Lloc,plmd.L_d);
}

__global__
void regularizelinekernel(struct_plmd plmd,real s)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  real deltax, dxds, L, dLds;
  __shared__ real Lloc[BLOCK];

  if (i<plmd.nx) {
    dxds=plmd.dxds_d[i];
    deltax=plmd.x_d[i]+s*dxds-plmd.xr_d[i];
    L=0.5*plmd.kx_d[i]*deltax*deltax;
    dLds=plmd.kx_d[i]*deltax*dxds;
  } else {
    L=0;
    dLds=0;
  }

  reduce(L,Lloc,plmd.L_d);
  reduce(dLds,Lloc,plmd.dLds_d);
}

__global__
void regularizedLdxkernel(struct_plmd plmd)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;

  if (i<plmd.nx) {
    plmd.dLdx_d[i]=plmd.kx_d[i]*(plmd.x_d[i]-plmd.xr_d[i]);
  }
}

void evaluateGimp(struct_plmd *plmd)
{
  if (PROFILE) {
    hipMemset(plmd->mc_Zprofile_d,0,plmd->nprof*NBINS*sizeof(real));
    // void profilekernel(struct_plmd plmd,real* lambda,real* inweight,real* weightprofile,real* outweight,real* Zprofile)
    profilekernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->mc_lambda_d,plmd->mc_ensweight_d,NULL,NULL,plmd->mc_Zprofile_d);
    // freeenergykernel<<<(plmd->nprof*NBINS+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->mc_Zprofile_d,plmd->Gimp_d,0,NULL,NULL,NULL);
    // Too noisy:
    // freeenergykernel<<<plmd->nprof,NBINS>>>(plmd[0],plmd->mc_Zprofile_d,plmd->Gimp_d,NULL,NULL);
    int s1,s2,i,j,k,kn;
    real *mc_Zprofile, *Gimp;
    mc_Zprofile=(real*)calloc(NBINS*plmd->nprof,sizeof(real));
    Gimp=(real*)calloc(NBINS,sizeof(real));
    hipMemcpy(mc_Zprofile,plmd->mc_Zprofile_d,NBINS*plmd->nprof*sizeof(real),hipMemcpyDeviceToHost);
    k=0;
    for (s1=0; s1<plmd->nsites; s1++) {
      for (s2=s1; s2<plmd->nsites; s2++) {
        if (s1==s2) { // Same site
          kn=k+plmd->nsubs[s1];
          for (i=0; i<NBINS; i++) {
            Gimp[i]=0;
            for (j=k; j<kn; j++) {
              Gimp[i]+=mc_Zprofile[NBINS*j+i];
            }
            Gimp[i]=-plmd->kT*log(Gimp[i]);
          }
          for (j=k; j<kn; j++) {
            hipMemcpy(&plmd->Gimp_d[NBINS*j],Gimp,NBINS*sizeof(real),hipMemcpyHostToDevice);
          }
          k=kn;

          kn=k+(plmd->nsubs[s1]*(plmd->nsubs[s1]-1))/2;
          for (i=0; i<NBINS; i++) {
            Gimp[i]=0;
            for (j=k; j<kn; j++) {
              Gimp[i]+=mc_Zprofile[NBINS*j+i];
            }
            Gimp[i]=-plmd->kT*log(Gimp[i]);
          }
          for (j=k; j<kn; j++) {
            hipMemcpy(&plmd->Gimp_d[NBINS*j],Gimp,NBINS*sizeof(real),hipMemcpyHostToDevice);
          }
          k=kn;

          if (plmd->nsubs[s1]>2) {
            kn=k+(plmd->nsubs[s1]*(plmd->nsubs[s1]-1))/2;
            for (i=0; i<NBINS; i++) {
              Gimp[i]=0;
              for (j=k; j<kn; j++) {
                Gimp[i]+=mc_Zprofile[NBINS*j+i];
              }
              Gimp[i]=-plmd->kT*log(Gimp[i]);
            }
            for (j=k; j<kn; j++) {
              hipMemcpy(&plmd->Gimp_d[NBINS*j],Gimp,NBINS*sizeof(real),hipMemcpyHostToDevice);
            }
            k=kn;
          }
        } else if (plmd->msprof) {
          kn=k+plmd->nsubs[s1]*plmd->nsubs[s2];
          for (i=0; i<NBINS; i++) {
            Gimp[i]=0;
            for (j=k; j<kn; j++) {
              Gimp[i]+=mc_Zprofile[NBINS*j+i];
            }
            Gimp[i]=-plmd->kT*log(Gimp[i]);
          }
          for (j=k; j<kn; j++) {
            hipMemcpy(&plmd->Gimp_d[NBINS*j],Gimp,NBINS*sizeof(real),hipMemcpyHostToDevice);
          }
          k=kn;
        }
      }
    }
    free(mc_Zprofile);
    free(Gimp);
  }

  if (MOMENT) {
    int i;
    real sum;
    sum=0;
    for (i=0; i<plmd->B; i++) {
      sum+=plmd->ensweight[i];
    }
    hipMemcpy(plmd->sumensweight_d,&sum,sizeof(real),hipMemcpyHostToDevice);
    hipMemset(plmd->moments_d,0,plmd->nbias*sizeof(real));
// void weightedenergykernel(struct_plmd plmd,real sign,real* lambda,real* weight,real* dEdx)
    weightedenergykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],-1,plmd->lambda_d,plmd->ensweight_d,plmd->moments_d);
  }
}

__global__
void likelihoodkernel(struct_plmd plmd,real s,real* L,real* dLds)
{
  if (L) {
    atomicAdd(L,plmd.Esum_d[0]/(plmd.sumensweight_d[0]*plmd.kT));
    if (s) atomicAdd(L,s*plmd.dEdssum_d[0]/(plmd.sumensweight_d[0]*plmd.kT));
    atomicAdd(L,log(plmd.mc_Z_d[0]));
  }
  if (dLds) {
    atomicAdd(dLds,plmd.dEdssum_d[0]/(plmd.sumensweight_d[0]*plmd.kT));
    atomicAdd(dLds,-plmd.mc_dEdssum_d[0]/(plmd.mc_Z_d[0]*plmd.kT));
  }
}

__global__
void gradientlikelihoodkernel(struct_plmd plmd,real* norm,real* dLdxin)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;

  if (i<plmd.nbias) {
    atomicAdd(&plmd.dLdx_d[i],-dLdxin[i]/(norm[0]*plmd.kT));
  }
}

void evaluateL(struct_plmd *plmd)
{
  /*{ // DEBUG
  FILE *fp;
  int i;
  fp=fopen("xbad.dat","r");
  for (i=0;i<plmd->nx;i++) {
    double buffer;
    fscanf(fp,"%lf",&buffer);
    plmd->x[i]=buffer;
  }
  fclose(fp);
  }*/

  hipMemcpy(plmd->x_d,plmd->x,plmd->nx*sizeof(real),hipMemcpyHostToDevice);
  hipMemset(plmd->L_d,0,sizeof(real));

  regularizeLkernel<<<(plmd->nx+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0]);

  energykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->x_d,plmd->lambda_d,plmd->E_d);

  // hipMemcpy(plmd->L,plmd->E_d,sizeof(real),hipMemcpyDeviceToHost); // DEBUG
  // fprintf(stderr,"Debug    energy[0]=%lg\n",plmd->L[0]); // DEBUG

  if (PROFILE) {
    hipMemset(plmd->Z_d,0,sizeof(real));
// void boltzmannkernel(struct_plmd plmd,real sign,real* energy,real s,real* denergyds,real* inweight,real* outweight,real* Z)
    boltzmannkernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>
      (plmd[0],-1,plmd->E_d,0,NULL,plmd->ensweight_d,plmd->weight_d,plmd->Z_d);

    hipMemset(plmd->Zprofile_d,0,plmd->nprof*NBINS*sizeof(real));
// void profilekernel(struct_plmd plmd,real* lambda,real* inweight,real* weightprofile,real* outweight,real* Zprofile)
    profilekernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->lambda_d,plmd->weight_d,NULL,NULL,plmd->Zprofile_d);
    freeenergykernel<<<plmd->nprof,NBINS>>>(plmd[0],plmd->Zprofile_d,plmd->G_d,plmd->L_d,NULL);
  }

  if (MOMENT) {
    energykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->x_d,plmd->mc_lambda_d,plmd->mc_E_d);
    hipMemset(plmd->Esum_d,0,sizeof(real));
    dotenergykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],1,plmd->ensweight_d,plmd->E_d,plmd->Esum_d);
    hipMemset(plmd->mc_Z_d,0,sizeof(real));
    boltzmannkernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>
      (plmd[0],1,plmd->mc_E_d,0,NULL,plmd->mc_ensweight_d,plmd->mc_weight_d,plmd->mc_Z_d);
    likelihoodkernel<<<1,1>>>(plmd[0],0,plmd->L_d,NULL);
  }

  hipMemcpy(plmd->L,plmd->L_d,sizeof(real),hipMemcpyDeviceToHost);

  fprintf(stdout,"New      L=%lg\n",plmd->L[0]);
}

void evaluateL_line(real s,struct_plmd *plmd)
{
  hipMemset(plmd->L_d,0,sizeof(real));
  hipMemset(plmd->dLds_d,0,sizeof(real));

  regularizelinekernel<<<(plmd->nx+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],s);

  if (PROFILE) {
    hipMemset(plmd->Z_d,0,sizeof(real));
// void boltzmannkernel(struct_plmd plmd,real sign,real* energy,real s,real* denergyds,real* inweight,real* outweight,real* Z)
    boltzmannkernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>
      (plmd[0],-1,plmd->E_d,s,plmd->dEds_d,plmd->ensweight_d,plmd->weight_d,plmd->Z_d);

    hipMemset(plmd->Zprofile_d,0,plmd->nprof*NBINS*sizeof(real));
// void profilekernel(struct_plmd plmd,real* lambda,real* inweight,real* weightprofile,real* outweight,real* Zprofile)
    profilekernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->lambda_d,plmd->weight_d,NULL,NULL,plmd->Zprofile_d);
    freeenergykernel<<<plmd->nprof,NBINS>>>(plmd[0],plmd->Zprofile_d,plmd->G_d,plmd->L_d,plmd->dLdZprofile_d);

    profilekernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->lambda_d,plmd->weight_d,plmd->dLdZprofile_d,plmd->dLdE_d,NULL);
    dotenergykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],-1,plmd->dLdE_d,plmd->dEds_d,plmd->dLds_d);
  }

  if (MOMENT) {
    hipMemset(plmd->mc_Z_d,0,sizeof(real));
    boltzmannkernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>
      (plmd[0],1,plmd->mc_E_d,s,plmd->mc_dEds_d,plmd->mc_ensweight_d,plmd->mc_weight_d,plmd->mc_Z_d);
    hipMemset(plmd->mc_dEdssum_d,0,sizeof(real));
    dotenergykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],1,plmd->mc_dEds_d,plmd->mc_weight_d,plmd->mc_dEdssum_d);
    likelihoodkernel<<<1,1>>>(plmd[0],s,plmd->L_d,plmd->dLds_d);
  }

  /*if (s==64) { // DEBUG
  printarray(plmd->dLdE_d,plmd->B,"dLdE.dat");
  printarray(plmd->dLds_d,1,"dLds.dat");
  printarray(plmd->Zprofile_d,NBINS*plmd->nprof,"Zprofile.dat");
  printarray(plmd->dLdZprofile_d,NBINS*plmd->nprof,"dLdZprofile.dat");
  }*/

  hipMemcpy(plmd->L,plmd->L_d,sizeof(real),hipMemcpyDeviceToHost);
  hipMemcpy(plmd->dLds,plmd->dLds_d,sizeof(real),hipMemcpyDeviceToHost);
}

void evaluatedLdx(struct_plmd *plmd)
{
  hipMemset(plmd->dLdx_d,0,plmd->nx*sizeof(real));

  regularizedLdxkernel<<<(plmd->nx+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0]);

  if (PROFILE) {
    freeenergykernel<<<plmd->nprof,NBINS>>>(plmd[0],plmd->Zprofile_d,plmd->G_d,NULL,plmd->dLdZprofile_d);
    profilekernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->lambda_d,plmd->weight_d,plmd->dLdZprofile_d,plmd->dLdE_d,NULL);

// void weightedenergykernel(struct_plmd plmd,real* lambda,real* weight,real* dEdx)
    weightedenergykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>
      (plmd[0],-1,plmd->lambda_d,plmd->dLdE_d,plmd->dLdx_d);
  }

  if (MOMENT) {
    hipMemset(plmd->mc_moments_d,0,plmd->nbias*sizeof(real));
// void weightedenergykernel(struct_plmd plmd,real sign,real* lambda,real* weight,real* dEdx)
    weightedenergykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],1,plmd->mc_lambda_d,plmd->mc_weight_d,plmd->mc_moments_d);

// void gradientlikelihoodkernel(struct_plmd plmd,real* norm,real* dLdxin)
    gradientlikelihoodkernel<<<(plmd->nbias+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->sumensweight_d,plmd->moments_d);
    gradientlikelihoodkernel<<<(plmd->nbias+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->mc_Z_d,plmd->mc_moments_d);
  }
  
  hipMemcpy(plmd->dLdx,plmd->dLdx_d,plmd->nx*sizeof(real),hipMemcpyDeviceToHost);
}

void resetHinv(struct_plmd *plmd)
{
  int i;
  // N^2 Hinv
  // for (i=0; i<plmd->nx*plmd->nx; i++) {
  //   plmd->Hinv[i]=0.0L;
  // }
  // for (i=0; i<plmd->nx; i++) {
  //   plmd->Hinv[(plmd->nx+1)*i]=1.0L;
  // }
  // N^1 Hinv
  for (i=0; i<plmd->nx; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }
}

void updateHinv(struct_plmd *plmd)
{
  /* // Begin N^2 Hinv
  int i,j;
  real DxDg,DgHinvDg;
  real c1,c2;

  DxDg=0.0L;
  for (i=0;i<plmd->nx;i++) {
    // Put Delta x and Delta dLdx in x0 and dLdx0, which hold previous values
    plmd->x0[i]=plmd->x[i]-plmd->x0[i];
    plmd->dLdx0[i]=plmd->dLdx[i]-plmd->dLdx0[i];
    DxDg+=plmd->x0[i]*plmd->dLdx0[i];
  }

  DgHinvDg=0.0L;
  for (i=0; i<plmd->nx; i++) {
    plmd->hi[i]=0.0L;
    for (j=0; j<plmd->nx; j++) {
      // put Hinv * Delta dLdx in hi (the search direction) as a buffer
      plmd->hi[i]+=plmd->Hinv[i*plmd->nx+j]*plmd->dLdx0[j];
    }
    DgHinvDg+=plmd->hi[i]*plmd->dLdx0[i];
  }
  c1=(1.0L+DgHinvDg/DxDg)/DxDg;
  c2=-1.0L/DxDg;

  for (i=0; i<plmd->nx; i++) {
    for (j=i; j<plmd->nx; j++) {
      plmd->Hinv[i*plmd->nx+j]+=c1*(plmd->x0[i]*plmd->x0[j])+c2*(plmd->hi[i]*plmd->x0[j]+plmd->x0[i]*plmd->hi[j]);
      plmd->Hinv[j*plmd->nx+i]=plmd->Hinv[i*plmd->nx+j];
    }
  }
  
  //   dx=xf-xi;
  //   dd=df-di;
  //   Hinv=Hinv+(1+(dd'*Hinv*dd)/(dx'*dd))*(dx*dx')/(dx'*dd)-((Hinv*dd*dx')+(Hinv*dd*dx')')/(dx'*dd);
  */ // End N^2 Hinv
  // Begin N^1 Hinv
  int i,j;

  if (plmd->Nmem<plmd->Nmemax) {
    plmd->Nmem++;
  }
  for (i=plmd->Nmem-1; i>0; i--) {
    for (j=0; j<plmd->nx; j++) {
      plmd->d_x[i*plmd->nx+j]=plmd->d_x[(i-1)*plmd->nx+j];
      plmd->d_dLdx[i*plmd->nx+j]=plmd->d_dLdx[(i-1)*plmd->nx+j];
    }
    plmd->rho[i]=plmd->rho[i-1];
  }

  plmd->rho[0]=0;
  for (i=0; i<plmd->nx; i++) {
    plmd->d_x[i]=plmd->x[i]-plmd->x0[i];
    plmd->d_dLdx[i]=plmd->dLdx[i]-plmd->dLdx0[i];
    plmd->rho[0]+=plmd->d_x[i]*plmd->d_dLdx[i];
  }
  plmd->rho[0]=1.0/plmd->rho[0];

  for (i=0; i<plmd->nx; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }
  // End N^1 Hinv
}

void projectHinv(struct_plmd *plmd)
{
/* // Begin N^2 Hinv
  int i,j;
  real dLds;
  //   hi=-Hinv*df;
  for (i=0; i<plmd->nx; i++) {
    plmd->hi[i]=0;
    for (j=0; j<plmd->nx; j++) {
      plmd->hi[i]+=-plmd->Hinv[i*plmd->nx+j]*plmd->dLdx[j];
    }
  }

  dLds=0;
  for (i=0; i<plmd->nx; i++) {
    dLds+=plmd->hi[i]*plmd->dLdx[i];
  }

  if (dLds>0) {
    fprintf(stderr,"Bad direction, reset Hinv\n");
    for (i=0; i<plmd->nx*plmd->nx; i++) {
      plmd->Hinv[i]=0;
    }
    for (i=0; i<plmd->nx; i++) {
      plmd->Hinv[i*(plmd->nx+1)]=1;
      plmd->hi[i]=plmd->dLdx[i];
    }
  }
*/ // End N^2 Hinv
// Begin N^1 Hinv
  int i,j;

  for (i=0; i<plmd->nx; i++) {
    plmd->hi[i]=plmd->dLdx[i];
  }
  for (i=0; i<plmd->Nmem; i++) {
    plmd->alpha[i]=0;
    for (j=0; j<plmd->nx; j++) {
      plmd->alpha[i]+=plmd->d_x[i*plmd->nx+j]*plmd->hi[j];
    }
    plmd->alpha[i]*=plmd->rho[i];
    for (j=0; j<plmd->nx; j++) {
      plmd->hi[j]+=-plmd->alpha[i]*plmd->d_dLdx[i*plmd->nx+j];
    }
  }
  /*
  // According to wikipedia, this is to ensure the step length is always about unity
  // https://en.wikipedia.org/wiki/Limited-memory_BFGS
  if (plmd->Nmem>0) {
    numer=0.0L
    denom=0.0L;
    for (i=0; i<plmd->nx; i++) {
      numer+=plmd->d_x[i]*plmd->hi[i];
      denom+=plmd->d_dLdx[i]*plmd->d_dLdx[i];
    }
    numer/=denom;
    for (i=0; i<plmd->nx; i++) {
      plmd->hi[i]=numer*plmd->d_dLdx[i]; // This seems like a horrible idea, maybe wikipedia has a typo...
    }
  }
  */
  for (i=plmd->Nmem-1; i>=0; i--) {
    plmd->beta[i]=0;
    for (j=0; j<plmd->nx; j++) {
      plmd->beta[i]+=plmd->d_dLdx[i*plmd->nx+j]*plmd->hi[j];
    }
    plmd->beta[i]*=plmd->rho[i];
    for (j=0; j<plmd->nx; j++) {
      plmd->hi[j]+=(plmd->alpha[i]-plmd->beta[i])*plmd->d_x[i*plmd->nx+j];
    }
  }

  for (i=0; i<plmd->nx; i++) {
    plmd->hi[i]*=-1;
  }
// End N^1 Hinv

  /*{ // DEBUG
  FILE *fp;
  int i;
  fp=fopen("dxdsbad.dat","r");
  for (i=0;i<plmd->nx;i++) {
    double buffer;
    fscanf(fp,"%lf",&buffer);
    plmd->hi[i]=buffer;
  }
  fclose(fp);
  }*/

  hipMemcpy(plmd->dxds_d,plmd->hi,plmd->nx*sizeof(real),hipMemcpyHostToDevice);

  energykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->dxds_d,plmd->lambda_d,plmd->dEds_d);
  if (MOMENT) {
    energykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],plmd->dxds_d,plmd->mc_lambda_d,plmd->mc_dEds_d);
    hipMemset(plmd->dEdssum_d,0,sizeof(real));
    dotenergykernel<<<(plmd->B+BLOCK-1)/BLOCK,BLOCK>>>(plmd[0],1,plmd->ensweight_d,plmd->dEds_d,plmd->dEdssum_d);
  }
}

void update_line(int step,struct_plmd *plmd)
{
  int i;
  real a,b,c,s;
  real s1,s2,s3;
  real L1,L2,L3;
  real dLds1,dLds2,dLds3;
  real L0;

  for (i=0; i<plmd->nx; i++) {
    plmd->x0[i]=plmd->x[i];
    plmd->dLdx0[i]=plmd->dLdx[i];
  }

  L0=plmd->L[0];

  s1=0.0;
  evaluateL_line(s1,plmd);
  L1=plmd->L[0];
  dLds1=plmd->dLds[0];
  if (dLds1>0) {
    fprintf(stdout,"Error, hi is pointing wrong way - halting\n");
    plmd->done=true;
    return;
    // exit(1);
  }
  
  s3=1.0;
  evaluateL_line(s3,plmd);
  L3=plmd->L[0];
  dLds3=plmd->dLds[0];

  while (dLds3<0 && s3<1e+8) {
    fprintf(stdout,"Seek %4d s=%lg %lg\n          L=%lg %lg\n       dLds=%lg %lg\n",
            step,(double) s1,(double) s3,
            (double) L1,(double) L3,
            (double) dLds1,(double) dLds3);
    s2=s1-dLds1*(s3-s1)/(dLds3-dLds1);
    s3=((1.5*s2>8*s3 || 1.5*s2<=0) ? 8*s3 : 1.5*s2); // s2 is expected 0. Go past it by 50%, unless that's an increase of more than a factor of 8.
    evaluateL_line(s3,plmd);
    L3=plmd->L[0];
    dLds3=plmd->dLds[0];
  }

  while (!isfinite(dLds3) && s3>1e-8) {
    fprintf(stdout,"Warning, overshot bound\n");
    fprintf(stdout,"Seek %4d s=%lg %lg\n          L=%lg %lg\n       dLds=%lg %lg\n",
            step,(double) s1,(double) s3,
            (double) L1,(double) L3,
            (double) dLds1,(double) dLds3);
    s3=0.95*s3;
    evaluateL_line(s3,plmd);
    L3=plmd->L[0];
    dLds3=plmd->dLds[0];
  }

  if (!(dLds3>0)) {
    fprintf(stdout,"Warning: Step %4d unsuccessful, halting minimization\n",step);
    fprintf(stdout,"Seek %4d s=%lg %lg\n          L=%lg %lg\n       dLds=%lg %lg\n",
            step,(double) s1,(double) s3,
            (double) L1,(double) L3,
            (double) dLds1,(double) dLds3);
    plmd->done=true;
    return;
  }

  /*if (s3>6) { // DEBUG
    for (i=0; i<10; i++) {
      evaluateL_line(0.1*i*s3,plmd);
      fprintf(stderr,"Debug   s=%g L=%g dLds=%g\n",0.1*i*s3,plmd->L[0],plmd->dLds[0]);
    }
  }*/

  /*if (!isfinite(dLds3)) { // DEBUG
    printarray(plmd->x_d,plmd->nx,"xbad.dat");
    printarray(plmd->dxds_d,plmd->nx,"dxdsbad.dat");
    exit(1);
  }*/

  s2=s1-dLds1*(s3-s1)/(dLds3-dLds1);
  evaluateL_line(s2,plmd);
  L2=plmd->L[0];
  dLds2=plmd->dLds[0];

  fprintf(stdout,"Step %4d s=%lg %lg %lg\n          L=%lg %lg %lg\n       dLds=%lg %lg %lg\n",
          step,(double) s1,(double) s2,(double) s3,
          (double) L1,(double) L2,(double) L3,
          (double) dLds1,(double) dLds2,(double) dLds3);

  for (i=0; i<15; i++) {
    if ((s2-s1)/s2<5e-7 || (s3-s2)/s2<5e-7 || dLds2==0) break;

    // Quadratic interpolation
    a=dLds1/((s1-s2)*(s1-s3));
    a+=dLds2/((s2-s1)*(s2-s3));
    a+=dLds3/((s3-s1)*(s3-s2));
    b=-dLds1*(s2+s3)/((s1-s2)*(s1-s3));
    b+=-dLds2*(s1+s3)/((s2-s1)*(s2-s3));
    b+=-dLds3*(s1+s2)/((s3-s1)*(s3-s2));
    c=dLds1*s2*s3/((s1-s2)*(s1-s3));
    c+=dLds2*s1*s3/((s2-s1)*(s2-s3));
    c+=dLds3*s1*s2/((s3-s1)*(s3-s2));
    s=(-b+sqrt(b*b-4*a*c))/(2*a);

    if (dLds2<0) {
      s1=s2;
      L1=L2;
      dLds1=dLds2;
    } else { // dLds2==0 already addressed above
      s3=s2;
      L3=L2;
      dLds3=dLds2;
    }

    if (s>s1 && s<s3) {
      // Use the earlier quadratic interpolation
      s2=s;
    } else {
      // Linear interpolation (secant method)
      fprintf(stdout,"Warning, fell back on linear interpolation\n");
      fprintf(stdout,"a=%lg b=%lg c=%lg s-=%lg s+=%lg s=%lg\n",(double)a,(double)b,(double)c,(double)((-b-sqrt(b*b-4*a*c))/(2*a)),(double)((-b+sqrt(b*b-4*a*c))/(2*a)),(double)s);
      s2=s1-dLds1*(s3-s1)/(dLds3-dLds1);
    }

    evaluateL_line(s2,plmd);
    L2=plmd->L[0];
    dLds2=plmd->dLds[0];

    fprintf(stdout,"Step %4d s=%lg %lg %lg\n          L=%lg %lg %lg\n       dLds=%lg %lg %lg\n",
            step,(double) s1,(double) s2,(double) s3,
            (double) L1,(double) L2,(double) L3,
            (double) dLds1,(double) dLds2,(double) dLds3);
  }

  fprintf(stdout,"Step %4d s=%lg %lg %lg\n          L=%lg %lg %lg\n       dLds=%lg %lg %lg\n",
          step,(double) s1,(double) s2,(double) s3,
          (double) L1,(double) L2,(double) L3,
          (double) dLds1,(double) dLds2,(double) dLds3);

  real stepLength2=0;
  real initGrad2=0;

  for (i=0; i<plmd->nx; i++) {
    plmd->x[i]=plmd->x0[i]+s2*plmd->hi[i];
    stepLength2+=(s2*plmd->hi[i])*(s2*plmd->hi[i]);
    initGrad2+=plmd->dLdx[i]*plmd->dLdx[i];
  }

  // fprintf(stderr,"Step %d smid1 %lg smid2 %lg L=%lg -> L1=%lg L2=%lg\n",step,(double) smid1,(double) smid2,(double) L0,(double) Lmid1,(double) Lmid2);
  fprintf(stdout,"Step %4d L=%24.16lf -> L2=%24.16lf, dL=%lg, step length=%lg\n",step,(double)L0,(double)L2,(double)(L2-L0),(double) sqrt(stepLength2));

  fprintf(plmd->fplog,"%24.16lf %24.16lf %lg %lg\n",(double)L0,(double)L2,(double)sqrt(initGrad2),(double)sqrt(stepLength2));

  if (sqrt(stepLength2)<5e-7) plmd->done=true;
  if (sqrt(stepLength2/plmd->nx)<plmd->criteria) { // criteria was 1e-2
    plmd->doneCount+=1;
    if (plmd->doneCount==2) plmd->done=true;
  } else {
    plmd->doneCount=0;
  }
}

real lineL(real s,struct_plmd *plmd)
{
  int i;

  for (i=0; i<plmd->nx; i++) {
    plmd->x[i]=plmd->x0[i]+s*plmd->hi[i];
  }

  evaluateL(plmd);
  return plmd->L[0];
}

void itterate(int step,struct_plmd *plmd)
{
  evaluateL(plmd);
  evaluatedLdx(plmd);

  if (step==0) {
    resetHinv(plmd);
  } else {
    updateHinv(plmd);
  }

  projectHinv(plmd);
  update_line(step,plmd);
}

void run(struct_plmd *plmd)
{
  int s;

  evaluateGimp(plmd);
  plmd->done=false;
  plmd->doneCount=0;
  for (s=0; s<250; s++) {
    itterate(s,plmd);
    if (plmd->done) break;
  }
}

void finish(struct_plmd *plmd,int argc, char *argv[])
{
  int i,j;
  FILE *fp;

  fp=fopen(argv[6],"w");
  for (i=0; i<plmd->nx; i++) {
    fprintf(fp," %lg",(double) plmd->x[i]);
  }
  fclose(fp);

  plmd->mc_weight=(real*)calloc(plmd->B,sizeof(real));
  hipMemcpy(plmd->mc_weight,plmd->mc_weight_d,plmd->B*sizeof(real),hipMemcpyDeviceToHost);
  hipMemcpy(plmd->mc_lambda,plmd->mc_lambda_d,plmd->B*plmd->nblocks*sizeof(real),hipMemcpyDeviceToHost);

  fp=fopen("mc_weight.dat","w");
  for (i=0; i<plmd->B; i++) {
    fprintf(fp," %lg\n",(double) plmd->mc_weight[i]);
  }
  fclose(fp);

  fp=fopen("mc_Lambda.dat","w");
  for (i=0; i<plmd->B; i++) {
    for (j=0; j<plmd->nblocks; j++) {
      fprintf(fp," %lg",(double) plmd->mc_lambda[i*plmd->nblocks+j]);
    }
    fprintf(fp,"\n");
  }
  fclose(fp);
  free(plmd->mc_weight);

  free(plmd->nsubs);
  free(plmd->block0);
  hipFree(plmd->block0_d);
  free(plmd->block2site);

  free(plmd->lambda);
  hipFree(plmd->lambda_d);
  free(plmd->ensweight);
  hipFree(plmd->ensweight_d);
  free(plmd->mc_lambda);
  hipFree(plmd->mc_lambda_d);
  free(plmd->mc_ensweight);
  hipFree(plmd->mc_ensweight_d);
  free(plmd->kx);
  hipFree(plmd->kx_d);
  free(plmd->xr);
  hipFree(plmd->xr_d);

  free(plmd->L);
  hipFree(plmd->L_d);
  free(plmd->dLds);
  hipFree(plmd->dLds_d);

  free(plmd->x);
  free(plmd->dLdx);
  hipFree(plmd->dLdx_d);
  // free(plmd->Hinv);
  free(plmd->d_x);
  free(plmd->d_dLdx);
  free(plmd->rho);
  free(plmd->alpha);
  free(plmd->beta);
  free(plmd->hi);
  free(plmd->x0);
  free(plmd->dLdx0);

  hipFree(plmd->E_d);
  hipFree(plmd->dEds_d);
  hipFree(plmd->mc_E_d);
  hipFree(plmd->mc_dEds_d);
  hipFree(plmd->weight_d);
  hipFree(plmd->mc_weight_d);
  hipFree(plmd->x_d);
  hipFree(plmd->dxds_d);
  hipFree(plmd->Z_d);
  hipFree(plmd->mc_Z_d);
  hipFree(plmd->Zprofile_d);
  hipFree(plmd->mc_Zprofile_d);
  hipFree(plmd->dLdZprofile_d);
  hipFree(plmd->mc_dLdZprofile_d);
  hipFree(plmd->dLdE_d);
  hipFree(plmd->Gimp_d);
  hipFree(plmd->G_d);

  hipFree(plmd->Esum_d);
  hipFree(plmd->dEdssum_d);
  hipFree(plmd->mc_dEdssum_d);
  hipFree(plmd->moments_d);
  hipFree(plmd->mc_moments_d);
  hipFree(plmd->sumensweight_d);

  fclose(plmd->fplog);

  free(plmd);
}

int main(int argc, char *argv[])
{
  struct_plmd *plmd;

  plmd = setup(argc,argv);
 
  run(plmd);

  finish(plmd,argc,argv);

  return 0;
}
